
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <assert.h>
#include <vector>
#include <queue>
#include <fstream>
#include <time.h>
 
#include <cmath>
#include <algorithm>
#include <string>
#include <set>
#include <climits>
#include <stack>
#include <sstream>
#include <chrono>

template <typename T> std::string tostr(const T& t)
{
    std::ostringstream os; os<<t; return os.str();
}
 
#define N 4
#define N2 16
#define STACK_LIMIT 64 * 8
#define MAX_CORE_NUM 15360
#define CORE_NUM 1536
// #define CORE_NUM 15360
// #define CORE_NUM 384
// #define CORE_NUM 192
// #define WARP_SIZE 8
// #define WARP_SIZE 4
#define WARP_SIZE 32
#define BLOCK_NUM 48
// #define BLOCK_NUM 480

using namespace std;

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}
 
static const int dx[4] = {0, -1, 0, 1};
static const int dy[4] = {1, 0, -1, 0};
// static const char dir[4] = {'r', 'u', 'l', 'd'}; 
static const int order[4] = {1, 0, 2, 3}; 
 
struct Node
{
    int puzzle[N2];
    int space;
    int md;
    int depth;
    int pre;
    bool operator < (const Node& n) const {
        return depth + md < n.depth + n.md;
    }

    bool operator > (const Node& n) const {
        return depth + md > n.depth + n.md;
    }
};

struct Lock {
    int *mutex;
    Lock( void ) {
        HANDLE_ERROR( hipMalloc( (void**)&mutex,
                              sizeof(int) ) );
        HANDLE_ERROR( hipMemset( mutex, 0, sizeof(int) ) );
    }

    ~Lock( void ) {
        hipFree( mutex );
    }

    __device__ void lock( void ) {
        while( atomicCAS( mutex, 0, 1 ) != 0 );
    __threadfence();
    }

    __device__ void unlock( void ) {
        __threadfence();
        atomicExch( mutex, 0 );
    }
};

Node s_node;
int tmp_md[N2*N2];
__constant__ int md[N2*N2];
int ans;
priority_queue<Node, vector<Node>, greater<Node> > pq;

int get_md_sum(int *puzzle) {
    int sum = 0;
    for (int i = 0; i < N2; ++i)
    {
        if(puzzle[i] == 0) continue;
        sum += tmp_md[i * N2 + puzzle[i]];
    }
    return sum;
}

void set_md() {
    for (int i = 0; i < N2; ++i)
    {
        for (int j = 0; j < N2; ++j)
        {
            tmp_md[i * N2 + j] = abs(i / N - j / N) + abs(i % N - j % N);
        }
    }
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(md), tmp_md, sizeof(int) * N2 * N2));
}

void input_table(char *input_file) {
    s_node = Node();
    fstream ifs(input_file);

    for (int i = 0; i < N2; ++i)
    {
        int tmp;
        // scanf("%d", &tmp);
        ifs >> tmp;
        // cin >> tmp;
        if(tmp == 0) {
            s_node.space = i;
        }
        s_node.puzzle[i] = tmp;
    }
    s_node.md = get_md_sum(s_node.puzzle);
    s_node.depth = 0;
    s_node.pre = -10;
}

bool create_root_set() {
    pq.push(s_node);
    while(pq.size() < BLOCK_NUM ) {
        Node cur_n = pq.top();
        pq.pop();
        if(cur_n.md == 0 ) {
            ans = cur_n.depth;
            return true;
        }
        int s_x = cur_n.space / N;
        int s_y = cur_n.space % N;
        for (int operator_order = 0; operator_order < 4; ++operator_order)
        {
            int i = order[operator_order];
            Node next_n = cur_n;
            int new_x = s_x + dx[i];
            int new_y = s_y + dy[i];
            if(new_x < 0  || new_y < 0 || new_x >= N || new_y >= N) continue; 
            if(max(cur_n.pre, i) - min(cur_n.pre, i) == 2) continue;
 
            //incremental manhattan distance
            next_n.md -= tmp_md[(new_x * N + new_y) * N2 + next_n.puzzle[new_x * N + new_y]];
            next_n.md += tmp_md[(s_x * N + s_y) * N2 + next_n.puzzle[new_x * N + new_y]];
 
            swap(next_n.puzzle[new_x * N + new_y], next_n.puzzle[s_x * N + s_y]);
            next_n.space = new_x * N + new_y;
            // assert(get_md_sum(new_n.puzzle) == new_n.md);
            next_n.depth++;
            next_n.pre = i;
            if(next_n.md == 0) {
                ans = next_n.depth;
                return true;
            }
            pq.push(next_n);
            if(pq.size() >= BLOCK_NUM){
                return false;
            }
        }
    }
    return false;
}

__global__ void dfs_kernel(int limit, Node *root_set, int *dev_flag, Lock *lock, int *loop_set) {
    __shared__ int shared_md[N2*N2];
    for (int i = threadIdx.x; i < N2*N2; i += blockDim.x)
    {
        shared_md[i] = md[i];
    }

    __syncthreads();

    __shared__ Node st[STACK_LIMIT];
    __shared__ int index;
    index = 0;
    st[0] = root_set[blockIdx.x];
    // index = WARP_SIZE / 4 - 1;
    // if(threadIdx.x % 4 == 0) {
    //     index++;
    // printf("stack index : %d  root node index %d\n", threadIdx.x / 4, blockIdx.x * (WARP_SIZE / 4 ) + threadIdx.x / 4);
    // st[threadIdx.x / 4] = root_set[blockIdx.x * (WARP_SIZE / 4 ) + threadIdx.x / 4];
    // }
    __syncthreads();

    int order[4] = {1, 0, 2, 3};
    int dx[4] = {0, -1, 0, 1};
    int dy[4] = {1, 0, -1, 0};

    int loop_count = 0;
    while(true) {
        bool stack_is_empty = (index <= -1);
        __syncthreads();
        if(stack_is_empty || *dev_flag != -1) break;

        Node cur_n;
        bool find_cur_n = false;
        int cur_n_idx = index - (threadIdx.x / 4);
        if(cur_n_idx >= 0) {
            cur_n = st[cur_n_idx];
            find_cur_n = true;
        }

        if(threadIdx.x == 0) {
            index = index >= (WARP_SIZE / 4 - 1) ? (index - WARP_SIZE / 4) : -1;
        }
        __syncthreads();


        if(find_cur_n) {
            if(cur_n.md == 0) {
                *dev_flag = cur_n.depth;
                goto LOOP;
                // return;
            }
            if(cur_n.depth + cur_n.md > limit) goto LOOP;
            int s_x = cur_n.space / N;
            int s_y = cur_n.space % N; 
            int operator_order = threadIdx.x % 4; 
            int i = order[operator_order];
            Node next_n = cur_n;
            int new_x = s_x + dx[i];
            int new_y = s_y + dy[i];
            if(new_x < 0  || new_y < 0 || new_x >= N || new_y >= N) goto LOOP; 
            if(max(cur_n.pre, i) - min(cur_n.pre, i) == 2) goto LOOP;

            //incremental manhattan distance
            next_n.md -= shared_md[(new_x * N + new_y) * N2 + next_n.puzzle[new_x * N + new_y]];
            next_n.md += shared_md[(s_x * N + s_y) * N2 + next_n.puzzle[new_x * N + new_y]];

            int a = next_n.puzzle[new_x * N + new_y];
            next_n.puzzle[new_x * N + new_y] = next_n.puzzle[s_x * N + s_y];
            next_n.puzzle[s_x * N + s_y] = a;

            next_n.space = new_x * N + new_y;
            // assert(get_md_sum(new_n.puzzle) == new_n.md);

            next_n.depth++;
            if(next_n.depth + next_n.md > limit) goto LOOP;
            next_n.pre = i;
            if(next_n.md == 0) {
                *dev_flag = next_n.depth;
                //return;
                goto LOOP;
            }
            for (int j = 0; j < WARP_SIZE; ++j)
            {
                if(j == threadIdx.x) {
                    // lock[blockIdx.x].lock();
                    atomicAdd(&index, 1);
                    // printf("%d:%d:%d\n", index, next_n.depth, next_n.pre);
                    st[index] = next_n;
                    // lock[blockIdx.x].unlock();
                }
            }

        }
        LOOP:
        loop_count++;
        __syncthreads();
    }
    loop_set[blockIdx.x] = loop_count; 
    return;
}


void divide_root_set(Node root, Node *new_root_set, int *new_root_set_index, int divide_num){
    priority_queue<Node, vector<Node>, greater<Node> > prq;
    prq.push(root);
    while(!prq.empty() && prq.size() < divide_num ) {
        Node cur_n = prq.top();
        prq.pop();
        if(cur_n.md == 0 ) {
            prq.push(cur_n);
            break;
        }
        int s_x = cur_n.space / N;
        int s_y = cur_n.space % N;
        for (int operator_order = 0; operator_order < 4; ++operator_order)
        {
            int i = order[operator_order];
            Node next_n = cur_n;
            int new_x = s_x + dx[i];
            int new_y = s_y + dy[i];
            if(new_x < 0  || new_y < 0 || new_x >= N || new_y >= N) continue; 
            if(max(cur_n.pre, i) - min(cur_n.pre, i) == 2) continue;
 
            //incremental manhattan distance
            next_n.md -= tmp_md[(new_x * N + new_y) * N2 + next_n.puzzle[new_x * N + new_y]];
            next_n.md += tmp_md[(s_x * N + s_y) * N2 + next_n.puzzle[new_x * N + new_y]];
 
            swap(next_n.puzzle[new_x * N + new_y], next_n.puzzle[s_x * N + s_y]);
            next_n.space = new_x * N + new_y;
            // assert(get_md_sum(new_n.puzzle) == new_n.md);
            next_n.depth++;
            next_n.pre = i;
            // if(next_n.md == 0) {
            //     prq.push(next_n);
            //     break;
            //     // ans = next_n.depth;
            //     // return true;
            // }
            prq.push(next_n);
        }
        // if(prq.size() >= divide_num){
        //     break
        //     while(prq.empty()) {
        //         new_root_set[*new_root_set_index] = prq.top();
        //         prq.pop();
        //         *new_root_set_index = *new_root_set_index + 1;
        //     }
        //     return;
        // }
    }
    while(!prq.empty()) {
        new_root_set[*new_root_set_index] = prq.top();
        prq.pop();
        *new_root_set_index = *new_root_set_index + 1;
    }
    return;
}

void ida_star() {
    pq = priority_queue<Node, vector<Node>, greater<Node> >();
    if(create_root_set()) {
        printf("%d\n", ans);
        return;
    }
    int root_node_size = pq.size();
    Node root_set[MAX_CORE_NUM];
    int i = 0;
    while(!pq.empty()) {
        Node n = pq.top();
        pq.pop();
        root_set[i] = n;
        i++;
    }

    int load_set[MAX_CORE_NUM];

    for (int limit = s_node.md; limit < 100; ++limit, ++limit)
    {
        int flag = -1;
        int *dev_flag;
        // int load;
        // int *dev_load;

        //gpu側で使う根集合のポインタ
        Node *dev_root_set;
        int *dev_load_set;
        //gpu側のメモリ割当て
        HANDLE_ERROR(hipMalloc((void**)&dev_root_set, root_node_size * sizeof(Node) ) );
        //root_setをGPU側のdev_root_setにコピー
        HANDLE_ERROR(hipMemcpy(dev_root_set, root_set, root_node_size * sizeof(Node), hipMemcpyHostToDevice) );


        //gpu側にメモリ割当
        HANDLE_ERROR(hipMalloc((void**)&dev_flag, sizeof(int)));
        hipMemcpy(dev_flag, &flag, sizeof(int), hipMemcpyHostToDevice);

        Lock    lock[BLOCK_NUM];
        Lock    *dev_lock;
        HANDLE_ERROR( hipMalloc( (void**)&dev_lock,
                              BLOCK_NUM * sizeof( Lock ) ) );
        HANDLE_ERROR( hipMemcpy( dev_lock, lock,
                              BLOCK_NUM * sizeof( Lock ),
                              hipMemcpyHostToDevice ) );
        HANDLE_ERROR(hipMalloc((void**)&dev_load_set, root_node_size * sizeof(int)));
        HANDLE_ERROR(hipMemset(dev_load_set, 0, root_node_size * sizeof(int)));

        // cout << root_node_size << endl;
        dfs_kernel<<<root_node_size, WARP_SIZE>>>(limit, dev_root_set, dev_flag, dev_lock, dev_load_set);


        HANDLE_ERROR(hipGetLastError());
        HANDLE_ERROR(hipDeviceSynchronize());
        HANDLE_ERROR(hipMemcpy(&flag, dev_flag, sizeof(int), hipMemcpyDeviceToHost));
        HANDLE_ERROR(hipMemcpy(&load_set, dev_load_set, root_node_size * sizeof(int), hipMemcpyDeviceToHost));

        HANDLE_ERROR(hipFree(dev_flag));
        HANDLE_ERROR(hipFree(dev_root_set));

        if(flag != -1) {
            cout << flag << endl;
            return;
        }

        int new_root_node_size = 0;
        Node new_root_set[MAX_CORE_NUM];

        //calculate load_balance
        int load_sum = 0;
        for (int i = 0; i < root_node_size; ++i)
        {
            load_sum += load_set[i];
            // cout << load_set[i] << " ";
        }
        // cout << endl;
        int load_av = load_sum / root_node_size;

        for (int i = 0; i < root_node_size; ++i)
        {
            int divide_num = load_av == 0 ? load_set[i] : (load_set[i]- 1) / load_av + 1;
            if(divide_num > 1) {
                divide_root_set(root_set[i], new_root_set, &new_root_node_size, divide_num);
            } else {
                new_root_set[new_root_node_size] = root_set[i];
                new_root_node_size++;
            }
            // cout << divide_num << " ";
        }
        // cout << "root_node_size:" <<root_node_size << endl;;

        assert(new_root_node_size <= MAX_CORE_NUM);


        for (int i = 0; i < new_root_node_size; ++i)
        {
            root_set[i] = new_root_set[i];
        }
        root_node_size = new_root_node_size;
    }
}

 
int main() {
    FILE *output_file;
    output_file = fopen("../result/korf100_block_parallel_result_with_staticlb_50.csv","w");

    set_md();
    for (int i = 0; i < 50; ++i)
    {
        string input_file = "../benchmarks/korf100/prob";
        if(i < 10) {
            input_file += "00";
        } else if(i < 100) {
            input_file += "0";
        }
        input_file += tostr(i);
        cout << input_file << " ";
        auto start = std::chrono::system_clock::now();

        input_table(const_cast<char*>(input_file.c_str()));
        ida_star();

        auto end = std::chrono::system_clock::now();
        auto diff = end - start;
        fprintf(output_file,"%f\n", std::chrono::duration_cast<std::chrono::nanoseconds>(diff).count() / (double)1000000000.0);
        // printf("%f\n", std::chrono::duration_cast<std::chrono::nanoseconds>(diff).count() / (double)1000000000.0);
    }
    fclose(output_file);
}
