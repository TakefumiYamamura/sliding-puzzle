
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <assert.h>
#include <vector>
#include <queue>
#include <fstream>
#include <time.h>
 
#include <cmath>
#include <algorithm>
#include <string>
#include <set>
#include <climits>
#include <stack>
#include <sstream>
#include <chrono>

// #define DEBUG
// #define DFS
// #define MANY_NODE

template <typename T> std::string tostr(const T& t)
{
    std::ostringstream os; os<<t; return os.str();
}
 
#define N 4
#define N2 16
#define STACK_LIMIT 64 * 12
#define MAX_CORE_NUM 100000
#define MAX_BLOCK_SIZE 64535
#define WARP_SIZE 32
#define THREAD_SIZE_PER_BLOCK 64
#define BLOCK_NUM 2048
// #define BLOCK_NUM 512
// #define BLOCK_NUM 4096
// #define BLOCK_NUM 48


using namespace std;

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}
 
static const int dx[4] = {0, -1, 0, 1};
static const int dy[4] = {1, 0, -1, 0};
// static const char dir[4] = {'r', 'u', 'l', 'd'}; 
static const int order[4] = {1, 0, 2, 3}; 
 
struct Node
{
    int puzzle[N2];
    int space;
    int md;
    int depth;
    int pre;
    bool operator < (const Node& n) const {
        return (depth + md) < (n.depth + n.md);
    }

    bool operator > (const Node& n) const {
        return (depth + md) > (n.depth + n.md);
    }
};

Node s_node;
int tmp_md[N2*N2];
__constant__ int md[N2*N2];
int ans;
priority_queue<Node, vector<Node>, greater<Node> > pq;
Node *global_st;

int get_md_sum(int *puzzle) {
    int sum = 0;
    for (int i = 0; i < N2; ++i)
    {
        if(puzzle[i] == 0) continue;
        sum += tmp_md[i * N2 + puzzle[i]];
    }
    return sum;
}


void set_md() {
    for (int i = 0; i < N2; ++i)
    {
        for (int j = 0; j < N2; ++j)
        {
            tmp_md[i * N2 + j] = abs(i / N - j / N) + abs(i % N - j % N);
        }
    }
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(md), tmp_md, sizeof(int) * N2 * N2));
}

void input_table(char *input_file) {
    s_node = Node();
    fstream ifs(input_file);

    for (int i = 0; i < N2; ++i)
    {
        int tmp;
        // scanf("%d", &tmp);
        ifs >> tmp;
        // cin >> tmp;
        if(tmp == 0) {
            s_node.space = i;
        }
        s_node.puzzle[i] = tmp;
    }
    s_node.md = get_md_sum(s_node.puzzle);
    s_node.depth = 0;
    s_node.pre = -10;
}

bool create_root_set() {
    pq.push(s_node);
    while(pq.size() < BLOCK_NUM ) {
        Node cur_n = pq.top();
        pq.pop();
        if(cur_n.md == 0 ) {
            ans = cur_n.depth;
            return true;
        }
        int s_x = cur_n.space / N;
        int s_y = cur_n.space % N;
        for (int operator_order = 0; operator_order < 4; ++operator_order)
        {
            int i = order[operator_order];
            Node next_n = cur_n;
            int new_x = s_x + dx[i];
            int new_y = s_y + dy[i];
            if(new_x < 0  || new_y < 0 || new_x >= N || new_y >= N) continue; 
            if(max(cur_n.pre, i) - min(cur_n.pre, i) == 2) continue;
 
            //incremental manhattan distance
            next_n.md -= tmp_md[(new_x * N + new_y) * N2 + next_n.puzzle[new_x * N + new_y]];
            next_n.md += tmp_md[(s_x * N + s_y) * N2 + next_n.puzzle[new_x * N + new_y]];
 
            swap(next_n.puzzle[new_x * N + new_y], next_n.puzzle[s_x * N + s_y]);
            next_n.space = new_x * N + new_y;
            // assert(get_md_sum(new_n.puzzle) == new_n.md);
            next_n.depth++;
            next_n.pre = i;
            if(next_n.md == 0) {
                ans = next_n.depth;
                return true;
            }
            pq.push(next_n);
        }
    }
    return false;
}

#ifdef MANY_NODE
__global__ void dfs_kernel(int limit, Node *root_set, int *dev_flag, int *loop_set, Node *global_st, int *dev_node_size) {

#else

__global__ void dfs_kernel(int limit, Node *root_set, int *dev_flag, int *loop_set, Node *global_st) {

#endif
    __shared__ int shared_md[N2*N2];
    for (int i = threadIdx.x; i < N2*N2; i += blockDim.x)
    {
        shared_md[i] = md[i];
    }

    __shared__ int mutex;
    mutex = 0;

    __syncthreads();

    __shared__ int index;
    #ifdef MANY_NODE
    index = 0;
    __syncthreads();
    int tmp_id = blockIdx.x * (THREAD_SIZE_PER_BLOCK / 4 ) + threadIdx.x / 4;
    if(threadIdx.x % 4 == 0 && *dev_node_size > tmp_id) {
        atomicAdd(&index, 1);
        //printf("stack index : %d  root node index %d\n", threadIdx.x / 4, blockIdx.x * (THREAD_SIZE_PER_BLOCK / 4 ) + threadIdx.x / 4);
        global_st[threadIdx.x / 4] = root_set[tmp_id];
    }

    #else
    index = 0;
    global_st[blockIdx.x * STACK_LIMIT + 0] = root_set[blockIdx.x];
    #endif

    __syncthreads();

    int order[4] = {1, 0, 2, 3};
    int dx[4] = {0, -1, 0, 1};
    int dy[4] = {1, 0, -1, 0};

    int loop_count = 0;
    while(true) {
        bool stack_is_empty = (index <= -1);
        __syncthreads();
        if(stack_is_empty || *dev_flag != -1) break;
        loop_count++;

        Node cur_n;
        bool find_cur_n = false;
        int cur_n_idx = index - (threadIdx.x / 4);
        if(cur_n_idx >= 0) {
            cur_n = global_st[blockIdx.x * STACK_LIMIT + cur_n_idx];
            assert(cur_n_idx < STACK_LIMIT);
            find_cur_n = true;
        }

        if(threadIdx.x == 0) {
            index = index >= (THREAD_SIZE_PER_BLOCK / 4 - 1) ? (index - THREAD_SIZE_PER_BLOCK / 4) : -1;
        }
        __syncthreads();

        Node next_n;

        if(find_cur_n) {
            if(cur_n.md == 0) {
                *dev_flag = cur_n.depth;
                goto LOOP;
            }
            if(cur_n.depth + cur_n.md > limit) goto LOOP;
            int s_x = cur_n.space / N;
            int s_y = cur_n.space % N; 
            int operator_order = threadIdx.x % 4; 
            int i = order[operator_order];
            next_n = cur_n;
            int new_x = s_x + dx[i];
            int new_y = s_y + dy[i];
            if(new_x < 0  || new_y < 0 || new_x >= N || new_y >= N) goto LOOP; 
            if(max(cur_n.pre, i) - min(cur_n.pre, i) == 2) goto LOOP;

            //incremental manhattan distance
            next_n.md -= shared_md[(new_x * N + new_y) * N2 + next_n.puzzle[new_x * N + new_y]];
            next_n.md += shared_md[(s_x * N + s_y) * N2 + next_n.puzzle[new_x * N + new_y]];

            int a = next_n.puzzle[new_x * N + new_y];
            next_n.puzzle[new_x * N + new_y] = next_n.puzzle[s_x * N + s_y];
            next_n.puzzle[s_x * N + s_y] = a;

            next_n.space = new_x * N + new_y;
            #ifdef DEBUG
            // int sum = 0;
            // for (int i = 0; i < N2; ++i)
            // {
            //     if(next_n.puzzle[i] == 0) continue;
            //     sum += shared_md[i * N2 + next_n.puzzle[i]];
            // }
            // assert(sum == next_n.md);
            #endif

            next_n.depth++;
            if(next_n.depth + next_n.md > limit) goto LOOP;
            next_n.pre = i;
            if(next_n.md == 0) {
                *dev_flag = next_n.depth;
                // for (int k = 0; k < N; ++k)
                // {
                //     for (int t = 0; t < N; ++t)
                //     {
                //         printf("%d ", next_n.puzzle[k*N + t] );
                //     }
                //     printf("\n");
                // }
                //return;
                goto LOOP;
            }
            for (int j = 0; j < WARP_SIZE; ++j)
            {
                if(j == (threadIdx.x % WARP_SIZE) ) {
                    while( atomicCAS(&mutex, 0, 1 ) != 0 );
                    index++;
                    global_st[blockIdx.x * STACK_LIMIT + index] = next_n;
                    atomicExch(&mutex, 0);
                    assert(index < STACK_LIMIT);
                }
            }
        }

        LOOP:
        __syncthreads();
    }
    loop_set[blockIdx.x] = loop_count; 
    return;
}


#ifndef DFS 
void divide_root_set(Node root, Node *new_root_set, int *new_root_set_index, int divide_num){
    priority_queue<Node, vector<Node>, greater<Node> > prq;
    // priority_queue<Node> prq;
    prq.push(root);
    while(!prq.empty() && prq.size() < divide_num ) {
        Node cur_n = prq.top();
        prq.pop();
        if(cur_n.md == 0 ) {
            prq.push(cur_n);
            // break;
        }
        int s_x = cur_n.space / N;
        int s_y = cur_n.space % N;
        for (int operator_order = 0; operator_order < 4; ++operator_order)
        {
            int i = order[operator_order];
            Node next_n = cur_n;
            int new_x = s_x + dx[i];
            int new_y = s_y + dy[i];
            if(new_x < 0  || new_y < 0 || new_x >= N || new_y >= N) continue; 
            if(max(cur_n.pre, i) - min(cur_n.pre, i) == 2) continue;
 
            //incremental manhattan distance
            next_n.md -= tmp_md[(new_x * N + new_y) * N2 + next_n.puzzle[new_x * N + new_y]];
            next_n.md += tmp_md[(s_x * N + s_y) * N2 + next_n.puzzle[new_x * N + new_y]];
 
            swap(next_n.puzzle[new_x * N + new_y], next_n.puzzle[s_x * N + s_y]);
            next_n.space = new_x * N + new_y;

            #ifdef DEBUG
            assert(get_md_sum(next_n.puzzle) == next_n.md);
            #endif

            next_n.depth++;
            next_n.pre = i;
            // if(next_n.md == 0) {
            //     prq.push(next_n);
            //     break;
            //     // ans = next_n.depth;
            //     // return true;
            // }
            prq.push(next_n);
        }
    }
    while(!prq.empty()) {
        new_root_set[*new_root_set_index] = prq.top();
        prq.pop();
        *new_root_set_index = *new_root_set_index + 1;
    }
    return;
}

#else

void divide_root_set(Node root, Node *new_root_set, int *new_root_set_index, int divide_num){
    stack<Node> st;
    st.push(root);
    while(!st.empty() && st.size() < divide_num ) {
        Node cur_n = st.top();
        st.pop();
        if(cur_n.md == 0 ) {
            st.push(cur_n);
            // break;
        }
        int s_x = cur_n.space / N;
        int s_y = cur_n.space % N;
        for (int operator_order = 0; operator_order < 4; ++operator_order)
        {
            int i = order[operator_order];
            Node next_n = cur_n;
            int new_x = s_x + dx[i];
            int new_y = s_y + dy[i];
            if(new_x < 0  || new_y < 0 || new_x >= N || new_y >= N) continue; 
            if(max(cur_n.pre, i) - min(cur_n.pre, i) == 2) continue;
 
            //incremental manhattan distance
            next_n.md -= tmp_md[(new_x * N + new_y) * N2 + next_n.puzzle[new_x * N + new_y]];
            next_n.md += tmp_md[(s_x * N + s_y) * N2 + next_n.puzzle[new_x * N + new_y]];
 
            swap(next_n.puzzle[new_x * N + new_y], next_n.puzzle[s_x * N + s_y]);
            next_n.space = new_x * N + new_y;
            #ifdef DEBUG
            assert(get_md_sum(next_n.puzzle) == next_n.md);
            #endif
            next_n.depth++;
            next_n.pre = i;
            st.push(next_n);
        }
    }
    while(!st.empty()) {
        new_root_set[*new_root_set_index] = st.top();
        st.pop();
        *new_root_set_index = *new_root_set_index + 1;
    }
    return;
}

#endif

Node root_set[MAX_BLOCK_SIZE];
Node new_root_set[MAX_BLOCK_SIZE];
int load_set[MAX_BLOCK_SIZE];
//メモリが足りなくなるのでグローバル変数として定義

void ida_star() {
    pq = priority_queue<Node, vector<Node>, greater<Node> >();
    if(create_root_set()) {
        printf("%d\n", ans);
        return;
    }
    int root_node_size = pq.size();
    int i = 0;
    while(!pq.empty()) {
        Node n = pq.top();
        pq.pop();
        root_set[i] = n;
        i++;
    }

    int flag = -1;
    int *dev_flag;
    Node *dev_root_set;
    int *dev_load_set;

    HANDLE_ERROR(hipMalloc((void**)&dev_root_set, MAX_BLOCK_SIZE * sizeof(Node) ) );
    HANDLE_ERROR(hipMalloc((void**)&dev_flag, sizeof(int)));
    hipMemcpy(dev_flag, &flag, sizeof(int), hipMemcpyHostToDevice);

    for (int limit = s_node.md; limit < 100; ++limit, ++limit)
    {
        #ifdef DEBUG
        auto start = std::chrono::system_clock::now();
        #endif

        HANDLE_ERROR(hipMemcpy(dev_root_set, root_set, root_node_size * sizeof(Node), hipMemcpyHostToDevice) );
        HANDLE_ERROR(hipMalloc((void**)&dev_load_set, root_node_size * sizeof(int)));
        HANDLE_ERROR(hipMemset(dev_load_set, 0, root_node_size * sizeof(int)));

        #ifdef DEBUG
        cout << "f_limit : " << limit << endl;
        cout << root_node_size << endl;
        #endif

        #ifdef MANY_NODE
        int *dev_node_size;
        HANDLE_ERROR(cudaMalloc((void**)&dev_node_size, sizeof(int)));
        cudaMemcpy(dev_node_size, &root_node_size, sizeof(int), cudaMemcpyHostToDevice);
        dfs_kernel<<<root_node_size / (THREAD_SIZE_PER_BLOCK / 4 ), THREAD_SIZE_PER_BLOCK>>>(limit, dev_root_set, dev_flag, dev_load_set, global_st, dev_node_size);
        #else
        dfs_kernel<<<root_node_size, THREAD_SIZE_PER_BLOCK>>>(limit, dev_root_set, dev_flag, dev_load_set, global_st);
        #endif

        HANDLE_ERROR(hipGetLastError());
        HANDLE_ERROR(hipDeviceSynchronize());
        HANDLE_ERROR(hipMemcpy(&flag, dev_flag, sizeof(int), hipMemcpyDeviceToHost));
        #ifdef MANY_NODE
        HANDLE_ERROR(cudaMemcpy(&load_set, dev_load_set, root_node_size / (THREAD_SIZE_PER_BLOCK / 4 ) * sizeof(int), cudaMemcpyDeviceToHost));
        #else
        HANDLE_ERROR(hipMemcpy(&load_set, dev_load_set, root_node_size * sizeof(int), hipMemcpyDeviceToHost));
        #endif

        #ifdef MANY_NODE
        HANDLE_ERROR(cudaFree(dev_node_size));
        #endif

        if(flag != -1) {
            cout << flag << endl;
            HANDLE_ERROR(hipFree(dev_flag));
            HANDLE_ERROR(hipFree(dev_root_set));
            HANDLE_ERROR(hipFree(dev_load_set));

            return;
        }

        int new_root_node_size = 0;

        //calculate load_balance
        int load_sum = 0;
        int max_load = 0;
        for (int i = 0; i < root_node_size; ++i)
        {
            load_sum += load_set[i];
            max_load = max(load_set[i], max_load);
            // cout << load_set[i] << " ";
        }
        // cout << "load sum " << load_sum << endl;
        int load_av = load_sum / root_node_size;
        #ifdef DEBUG
        cout << "load average " << load_av << endl;
        cout << "max load " << max_load << endl;
        int stat_cnt[10] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
        #endif
        for (int i = 0; i < root_node_size; ++i)
        {
            #ifdef DEBUG
            if (load_set[i] < load_av)
                stat_cnt[0]++;
            else if (load_set[i] < 2 * load_av)
                stat_cnt[1]++;
            else if (load_set[i] < 4 * load_av)
                stat_cnt[2]++;
            else if (load_set[i] < 8 * load_av)
                stat_cnt[3]++;
            else if (load_set[i] < 16 * load_av)
                stat_cnt[4]++;
            else if (load_set[i] < 32 * load_av)
                stat_cnt[5]++;
            else if (load_set[i] < 64 * load_av)
                stat_cnt[6]++;
            else if (load_set[i] < 128 * load_av)
                stat_cnt[7]++;
            else
                stat_cnt[8]++;
            #endif
            int divide_num = load_av == 0 ? load_set[i] : (load_set[i]- 1) / load_av + 1;

            if((divide_num > 1 && new_root_node_size + root_node_size - i + divide_num < MAX_BLOCK_SIZE/2) || (divide_num > 2 && new_root_node_size + root_node_size - i + divide_num < MAX_BLOCK_SIZE)) {
                #ifdef DEBUG
                int tmp = new_root_node_size;
                #endif
                divide_root_set(root_set[i], new_root_set, &new_root_node_size, divide_num);
                #ifdef DEBUG
                // cout << tmp << " " << new_root_node_size << endl;
                assert(tmp <= new_root_node_size);
                assert(new_root_node_size < MAX_BLOCK_SIZE);
                #endif
            } else {
                new_root_set[new_root_node_size] = root_set[i];
                new_root_node_size++;
            }

        }
        #ifdef DEBUG
        printf("STAT: distr: av=%d, 2av=%d, 4av=%d, 8av=%d, 16av=%d, 32av=%d, "
             "64av=%d, 128av=%d, more=%d\n",
             stat_cnt[0], stat_cnt[1], stat_cnt[2], stat_cnt[3], stat_cnt[4],
             stat_cnt[5], stat_cnt[6], stat_cnt[7], stat_cnt[8]);
        cout << "root_node_size:" << root_node_size << endl;
        cout << "new_root_node_size:" << new_root_node_size << endl;
        auto end = std::chrono::system_clock::now();
        auto diff = end - start;
        printf("executed time is %f\n", std::chrono::duration_cast<std::chrono::nanoseconds>(diff).count() / (double)1000000000.0);
        cout << "------" << endl;
        cout << endl;
        #endif

        assert(new_root_node_size <= MAX_CORE_NUM);


        for (int i = 0; i < new_root_node_size; ++i)
        {
            root_set[i] = new_root_set[i];
        }
        root_node_size = new_root_node_size;
    }
}

 
int main() {
    int problems_num = 100;
    #ifndef DEBUG
    FILE *output_file;
    string output_file_str = "../result/korf100_block_parallel_result_with_staticlb_global" + tostr(problems_num) + "_" + tostr(BLOCK_NUM) + "_" + tostr(THREAD_SIZE_PER_BLOCK) + ".csv";
    output_file = fopen(const_cast<char*>(output_file_str.c_str()),"w");
    #endif

    HANDLE_ERROR(hipMalloc((void**)&global_st, MAX_BLOCK_SIZE * STACK_LIMIT * sizeof(Node) ) );

    set_md();
    for (int i = 0; i < problems_num; ++i)
    {
        string input_file = "../benchmarks/korf100/prob";
        if(i < 10) {
            input_file += "00";
        } else if(i < 100) {
            input_file += "0";
        }
        input_file += tostr(i);
        cout << input_file << " ";
        auto start = std::chrono::system_clock::now();

        input_table(const_cast<char*>(input_file.c_str()));
        ida_star();

        auto end = std::chrono::system_clock::now();
        auto diff = end - start;
        #ifndef DEBUG
        fprintf(output_file,"%f\n", std::chrono::duration_cast<std::chrono::nanoseconds>(diff).count() / (double)1000000000.0);
        #endif
        #ifdef DEBUG
        printf("%f\n", std::chrono::duration_cast<std::chrono::nanoseconds>(diff).count() / (double)1000000000.0);
        printf("thread per block : %d\n", THREAD_SIZE_PER_BLOCK);
        #endif
    }
    HANDLE_ERROR(hipFree(global_st));

    #ifndef DEBUG
    fclose(output_file);
    #endif
}
