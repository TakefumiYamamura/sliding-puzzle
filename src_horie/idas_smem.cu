
#include <hip/hip_runtime.h>
#include <stdbool.h>

#define PACKED
#define SEARCH_ALL_THE_BEST
#define COLLECT_LOG
#undef USE_PRECOMPUTED_HDIFF

#define BLOCK_DIM (32)
#define N_INIT_DISTRIBUTION (BLOCK_DIM * 64)
#define MAX_GPU_PLAN (24)
#define MAX_BUF_RATIO (256)

#define STATE_WIDTH 4
#define STATE_N (STATE_WIDTH * STATE_WIDTH)

typedef unsigned char uchar;
typedef signed char   Direction;
#define dir_reverse(dir) ((Direction)(3 - (dir)))
#define DIR_N 4
#define DIR_FIRST 0
#define DIR_UP 0
#define DIR_RIGHT 1
#define DIR_LEFT 2
#define DIR_DOWN 3
#define POS_X(pos) ((pos) &3)
#define POS_Y(pos) ((pos) >> 2)

typedef struct search_stat_tag
{
    bool                   solved;
    int                    len;
    unsigned long long int loads;
#ifdef COLLECT_LOG
	unsigned long long int nodes_expanded;
#endif
	//bool assert_failed;
} search_stat;
typedef struct input_tag
{
    uchar     tiles[STATE_N];
    int       init_depth;
    Direction parent_dir;
} Input;

/* state implementation */

/*
 * goal: [0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15]
 */

#ifdef USE_PRECOMPUTED_HDIFF
__device__ __shared__ static signed char h_diff_table_shared[STATE_N][STATE_N] [DIR_N];
#endif

typedef struct state_tag
{
#ifndef PACKED
    uchar tile[STATE_N];
#else
    unsigned long long tile;
#endif
    uchar     empty;
    uchar     depth;
    Direction parent_dir;
    uchar     h_value; /* ub of h_value is STATE_WIDTH*2*(STATE_N-1), e.g. 90 */
} d_State;

#ifndef PACKED
#define state_tile_get(i) (state->tile[i])
#define state_tile_set(i, v) (state->tile[i] = (v))

#else
#define STATE_TILE_BITS 4
#define STATE_TILE_MASK ((1ull << STATE_TILE_BITS) - 1)
#define state_tile_ofs(i) (i << 2)
#define state_tile_get(i)                                                      \
    ((state->tile & (STATE_TILE_MASK << state_tile_ofs(i))) >>                 \
     state_tile_ofs(i))
#define state_tile_set(i, val)                                                 \
    do                                                                         \
    {                                                                          \
        state->tile &= ~((STATE_TILE_MASK) << state_tile_ofs(i));              \
        state->tile |= ((unsigned long long) val) << state_tile_ofs(i);        \
    } while (0)
#endif

#define distance(i, j) ((i) > (j) ? (i) - (j) : (j) - (i))
__device__ static void
state_init(d_State *state, Input *input)
{
    state->depth      = input->init_depth;
    state->parent_dir = input->parent_dir;
    for (int i = 0; i < STATE_N; ++i)
    {
        if (input->tiles[i] == 0)
            state->empty = i;
        state_tile_set(i, input->tiles[i]);
    }

    state->h_value = 0;
    for (int i = 0; i < STATE_N; ++i)
    {
	    uchar tile = state_tile_get(i);
	    if (tile == 0)
		    continue;
	    state->h_value += distance(POS_X(i), POS_X(tile));
	    state->h_value += distance(POS_Y(i), POS_Y(tile));
    }
}

__device__ static inline bool
state_is_goal(d_State state)
{
    return state.h_value == 0;
}

__device__ static inline int
state_get_f(d_State state)
{
    return state.depth + state.h_value;
}

__device__ __shared__ static bool movable_table_shared[STATE_N][DIR_N];

__device__ static inline bool
state_movable(d_State state, Direction dir)
{
    return movable_table_shared[state.empty][dir];
}

__device__ __constant__ const static int pos_diff_table[DIR_N] = {
    -STATE_WIDTH, 1, -1, +STATE_WIDTH};

__device__ static inline int
calc_h_diff(int opponent, int from, int rev_dir)
{
	int goal_x = POS_X(opponent), goal_y = POS_Y(opponent);
	int from_x = POS_X(from), from_y = POS_Y(from);
	if (rev_dir == DIR_LEFT)
		return goal_x > from_x ? -1 : 1;
	else if (rev_dir == DIR_RIGHT)
		return goal_x < from_x ? -1 : 1;
	else if (rev_dir == DIR_UP)
		return goal_y > from_y ? -1 : 1;
	else
		return goal_y < from_y ? -1 : 1;
}

__device__ static inline void
state_move(d_State *state, Direction dir)
{
    int new_empty = state->empty + pos_diff_table[dir];
    int opponent  = state_tile_get(new_empty);

#ifdef USE_PRECOMPUTED_HDIFF
    state->h_value += h_diff_table_shared[opponent][new_empty][dir];
#else
    state->h_value += calc_h_diff(opponent, new_empty, dir);
#endif
    state_tile_set(state->empty, opponent);
    state->empty      = new_empty;
    state->parent_dir = dir;
    ++state->depth;
}

/* stack implementation */
#define STACK_BUF_LEN (MAX_GPU_PLAN * (BLOCK_DIM/DIR_N))

typedef struct div_stack_tag
{
    unsigned int n;
    d_State      buf[STACK_BUF_LEN];
} d_Stack;

__device__ static inline bool
stack_is_empty(d_Stack *stack)
{
	bool ret = (stack->n == 0);
	__syncthreads();
	return ret;
}

__device__ static inline void
stack_put(d_Stack *stack, d_State *state, bool put)
{
	if (put)
	{
		unsigned int i = atomicInc( &stack->n, UINT_MAX); /* slow? especially in old CC environment */
		stack->buf[i] = *state;
	}
	__syncthreads();
}
__device__ static inline bool
stack_pop(d_Stack *stack, d_State *state)
{
    int tid = threadIdx.x;
    int i   = (int) stack->n - 1 - (int) (tid >> 2);
    if (i >= 0)
        *state = stack->buf[i];
    __syncthreads();
    if (tid == 0)
        stack->n = stack->n >= BLOCK_DIM / DIR_N ?
			stack->n - BLOCK_DIM / DIR_N : 0;
	__syncthreads();
    return i >= 0;
}

//__device__ __shared__ Direction candidate_dir_table[4][3] = {}

/*
 * solver implementation
 */
__device__ static void
idas_internal(d_Stack *stack, int f_limit, search_stat *stat)
{
	d_State state;
    unsigned long long int loop_cnt = 0;
#ifdef COLLECT_LOG
    unsigned long long int nodes_expanded = 0;
#endif
	if (threadIdx.x == 0)
		stat->solved = false;

    for (;;)
    {
        if (stack_is_empty(stack))
		{
			stat->loads = loop_cnt;
#ifdef COLLECT_LOG
			atomicAdd(&stat->nodes_expanded, nodes_expanded);
#endif
			break;
		}

        ++loop_cnt;
        bool found = stack_pop(stack, &state),
			 put = false;

        if (found)
        {
            Direction dir = threadIdx.x & 3;
#ifdef COLLECT_LOG
			nodes_expanded++;
#endif

			/* NOTE: candidate_dir_table may be effective to avoid divergence */
            if (state.parent_dir == dir_reverse(dir))
                continue;

            if (state_movable(state, dir))
            {
                state_move(&state, dir);

                if (state_get_f(state) <= f_limit)
                {
                    if (state_is_goal(state))
					{
#ifndef SEARCH_ALL_THE_BEST
						asm("trap;");
#else
						stat->loads = loop_cnt;
						stat->len = state.depth;
						stat->solved = true;
#endif

#ifdef COLLECT_LOG
						atomicAdd(&stat->nodes_expanded, nodes_expanded);
#endif
					}
                    else
                        put = true;
                }
            }
        }

		stack_put(stack, &state, put);
    }
}

/* XXX: movable table is effective in this case? */
__global__ void
idas_kernel(Input *input, search_stat *stat, int f_limit,
            signed char *h_diff_table, bool *movable_table)
{
    __shared__ d_Stack     stack;
    int tid = threadIdx.x;
	int bid = blockIdx.x;
	if (tid == 0)
		stat[bid].loads = 0;

	d_State state;
	state_init(&state, &input[bid]);
	if (state_get_f(state) > f_limit)
		return;

	if (tid == 0)
	{
		stack.buf[0] = state;
		stack.n      = 1;
	}

    for (int i = tid; i < STATE_N * DIR_N; i += blockDim.x)
        if (i < STATE_N * DIR_N)
            movable_table_shared[i / DIR_N][i % DIR_N] = movable_table[i];

#ifdef USE_PRECOMPUTED_HDIFF
    for (int dir = 0; dir < DIR_N; ++dir)
        for (int j = tid; j < STATE_N * STATE_N; j += blockDim.x)
            if (j < STATE_N * STATE_N)
                h_diff_table_shared[j / STATE_N][j % STATE_N][dir] =
                    h_diff_table[j * DIR_N + dir];
#endif

	__syncthreads();
    idas_internal(&stack, f_limit, &stat[bid]);
}

/* host library implementation */

#include <errno.h>
#include <limits.h>
#include <stddef.h>
#include <stdio.h>
#include <stdlib.h>

#ifndef UNABLE_LOG
#define elog(...) fprintf(stderr, __VA_ARGS__)
#else
#define elog(...) ;
#endif

void *
palloc(size_t size)
{
    void *ptr = malloc(size);
    if (!ptr)
        elog("malloc failed\n");

    return ptr;
}

void *
repalloc(void *old_ptr, size_t new_size)
{
    void *ptr = realloc(old_ptr, new_size);
    if (!ptr)
        elog("realloc failed\n");

    return ptr;
}

void
pfree(void *ptr)
{
    if (!ptr)
        elog("empty ptr\n");
    free(ptr);
}

#include <assert.h>
#include <stdbool.h>
#include <stdlib.h>
#include <string.h>

typedef unsigned char idx_t;
/*
 *  [0,0] [1,0] [2,0] [3,0]
 *  [0,1] [1,1] [2,1] [3,1]
 *  [0,2] [1,2] [2,2] [3,2]
 *  [0,3] [1,3] [2,3] [3,3]
 */

/*
 * goal state is
 * [0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15]
 */

typedef struct state_tag_cpu
{
    int       depth; /* XXX: needed? */
    uchar     pos[STATE_WIDTH][STATE_WIDTH];
    idx_t     i, j; /* pos of empty */
    Direction parent_dir;
    int       h_value;
} * State;

#define v(state, i, j) ((state)->pos[i][j])
#define ev(state) (v(state, state->i, state->j))
#define lv(state) (v(state, state->i - 1, state->j))
#define dv(state) (v(state, state->i, state->j + 1))
#define rv(state) (v(state, state->i + 1, state->j))
#define uv(state) (v(state, state->i, state->j - 1))

static uchar from_x[STATE_WIDTH * STATE_WIDTH],
    from_y[STATE_WIDTH * STATE_WIDTH];

static inline void
fill_from_xy(State from)
{
    for (idx_t x = 0; x < STATE_WIDTH; ++x)
        for (idx_t y = 0; y < STATE_WIDTH; ++y)
        {
            from_x[v(from, x, y)] = x;
            from_y[v(from, x, y)] = y;
        }
}

static inline int
heuristic_manhattan_distance(State from)
{
    int h_value = 0;

    fill_from_xy(from);

    for (idx_t i = 1; i < STATE_N; ++i)
    {
        h_value += distance(from_x[i], i & 3);
        h_value += distance(from_y[i], i >> 2);
    }

    return h_value;
}

bool
state_is_goal(State state)
{
    return state->h_value == 0;
}

static inline State
state_alloc(void)
{
    return (State) palloc(sizeof(struct state_tag_cpu));
}

static inline void
state_free(State state)
{
    pfree(state);
}

State
state_init(uchar v_list[STATE_WIDTH * STATE_WIDTH], int init_depth)
{
    State state = state_alloc();
    int   cnt   = 0;

    state->depth      = init_depth;
    state->parent_dir = (Direction) -1;

    for (idx_t j = 0; j < STATE_WIDTH; ++j)
        for (idx_t i = 0; i < STATE_WIDTH; ++i)
        {
            if (v_list[cnt] == 0)
            {
                state->i = i;
                state->j = j;
            }
            v(state, i, j) = v_list[cnt++];
        }

    state->h_value = heuristic_manhattan_distance(state);

    return state;
}

void
state_fini(State state)
{
    state_free(state);
}

State
state_copy(State src)
{
    State dst = state_alloc();

    memcpy(dst, src, sizeof(*src));

    return dst;
}

static inline bool
state_left_movable(State state)
{
    return state->i != 0;
}
static inline bool
state_down_movable(State state)
{
    return state->j != STATE_WIDTH - 1;
}
static inline bool
state_right_movable(State state)
{
    return state->i != STATE_WIDTH - 1;
}
static inline bool
state_up_movable(State state)
{
    return state->j != 0;
}

bool
state_movable(State state, Direction dir)
{
    return (dir != DIR_LEFT || state_left_movable(state)) &&
           (dir != DIR_DOWN || state_down_movable(state)) &&
           (dir != DIR_RIGHT || state_right_movable(state)) &&
           (dir != DIR_UP || state_up_movable(state));
}

#define h_diff(who, from_i, from_j, dir)                                       \
    (h_diff_table[((who) << 6) + ((from_j) << 4) + ((from_i) << 2) + (dir)])
static int h_diff_table[STATE_N * STATE_N * DIR_N] = {
    1,  1,  1,  1,  1,  1,  -1, 1,  1,  1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  1,
    1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  1,  1,  -1, 1,
    -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  -1,
    1,  -1, 1,  -1, 1,  -1, 1,  1,  -1, 1,  1,  1,  1,  1,  1,  1,  1,  -1, 1,
    1,  1,  -1, 1,  -1, -1, 1,  1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  -1, 1,  -1,
    1,  -1, -1, 1,  1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, -1,
    1,  1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  1,  -1, 1,  1,  1,
    -1, 1,  1,  1,  1,  1,  1,  1,  1,  -1, 1,  -1, -1, 1,  1,  -1, -1, 1,  1,
    -1, 1,  1,  1,  -1, 1,  -1, 1,  -1, -1, 1,  1,  -1, -1, 1,  1,  -1, 1,  1,
    1,  -1, 1,  -1, 1,  -1, -1, 1,  1,  -1, -1, 1,  1,  -1, 1,  1,  1,  -1, 1,
    -1, 1,  1,  -1, 1,  1,  1,  -1, 1,  1,  1,  -1, 1,  1,  1,  1,  1,  1,  -1,
    -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,  -1, 1,  1,  1,  -1, -1, 1,  1,
    -1, -1, 1,  1,  -1, -1, 1,  1,  -1, 1,  1,  1,  -1, -1, 1,  1,  -1, -1, 1,
    1,  -1, -1, 1,  1,  -1, 1,  1,  1,  1,  1,  1,  -1, 1,  1,  -1, -1, 1,  1,
    -1, -1, 1,  1,  -1, -1, 1,  1,  1,  1,  1,  1,  -1, 1,  1,  1,  -1, 1,  1,
    1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,
    -1, 1,  1,  1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  1,  -1, 1,
    -1, 1,  1,  1,  -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,  -1, 1,  1,  1,  1,
    1,  1,  1,  1,  -1, 1,  1,  1,  -1, 1,  -1, -1, 1,  1,  -1, 1,  1,  1,  -1,
    1,  -1, 1,  -1, 1,  -1, 1,  -1, -1, 1,  1,  -1, 1,  1,  1,  -1, 1,  -1, 1,
    -1, 1,  -1, 1,  1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  1,  -1,
    -1, 1,  -1, 1,  1,  1,  -1, 1,  1,  1,  1,  1,  1,  1,  1,  -1, 1,  -1, -1,
    1,  1,  -1, -1, 1,  1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  -1, -1, 1,  1,  -1,
    -1, 1,  1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  1,  -1, 1,  -1, 1,  -1, 1,  -1,
    1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  1,  1,  -1, 1,
    1,  1,  1,  1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,  -1, 1,
    1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,  -1, 1,  1,  1,  1,
    1,  1,  -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,  1,  -1,
    1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,  1,  1,  1,  1,  -1,
    1,  1,  1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  -1, 1,
    -1, 1,  -1, 1,  -1, 1,  1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  1,  -1, -1, 1,
    1,  -1, -1, 1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  1,  -1, -1, 1,  1,  -1, -1,
    1,  -1, 1,  1,  1,  1,  1,  1,  1,  1,  -1, 1,  1,  1,  -1, 1,  -1, -1, 1,
    1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  1,  -1, 1,  -1, 1,  -1,
    1,  -1, 1,  1,  1,  -1, 1,  1,  -1, -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,
    1,  1,  -1, 1,  1,  -1, -1, 1,  -1, 1,  1,  1,  -1, 1,  1,  1,  1,  1,  1,
    1,  1,  -1, 1,  -1, -1, 1,  1,  -1, -1, 1,  1,  -1, 1,  1,  1,  -1, 1,  -1,
    1,  1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  -1,
    1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  1,  1,
    -1, 1,  1,  1,  -1, 1,  1,  1,  1,  1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,
    -1, -1, 1,  1,  -1, 1,  1,  1,  1,  1,  1,  -1, 1,  1,  -1, -1, 1,  1,  -1,
    -1, 1,  1,  -1, -1, 1,  1,  1,  -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,
    -1, -1, 1,  1,  1,  -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,
    1,  1,  1,  1,  1,  -1, 1,  1,  1,  -1, 1,  1,  1,  -1, 1,  1,  -1, 1,  -1,
    1,  1,  1,  -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,  -1, 1,  -1, 1,  1,  1,
    -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  1,
    -1, -1, 1,  1,  -1, -1, 1,  -1, 1,  1,  1,  1,  1,  1,  1,  1,  -1, 1,  1,
    1,  -1, 1,  1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  1,  -1, -1,
    1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  1,  -1, -1, 1,  -1, 1,
    -1, 1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  1,  -1, -1, 1,  -1, 1,  1,  1,  -1,
    1,  1,  1,  1,  1,  1,  1,  1,  -1, 1,  1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,
    -1, 1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1,
    1,  1,  1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  1,  1,
    -1, 1,  -1, 1,  1,  1,  -1, 1,  1,  1,  -1, 1,  1,  1,  1,  1,  1};

void
state_move(State state, Direction dir)
{
    idx_t who;
    assert(state_movable(state, dir));

    switch (dir)
    {
    case DIR_LEFT:
        who = ev(state) = lv(state);
        state->i--;
        break;
    case DIR_DOWN:
        who = ev(state) = dv(state);
        state->j++;
        break;
    case DIR_RIGHT:
        who = ev(state) = rv(state);
        state->i++;
        break;
    case DIR_UP:
        who = ev(state) = uv(state);
        state->j--;
        break;
    default:
        elog("unexpected direction");
        assert(false);
    }

    state->h_value =
        state->h_value + h_diff(who, state->i, state->j, dir_reverse(dir));
    state->parent_dir = dir;
}

bool
state_pos_equal(State s1, State s2)
{
    for (idx_t i = 0; i < STATE_WIDTH; ++i)
        for (idx_t j = 0; j < STATE_WIDTH; ++j)
            if (v(s1, i, j) != v(s2, i, j))
                return false;

    return true;
}

size_t
state_hash(State state)
{
    size_t hash_value = 0;
    for (idx_t i = 0; i < STATE_WIDTH; ++i)
        for (idx_t j = 0; j < STATE_WIDTH; ++j)
            hash_value ^= (v(state, i, j) << ((i * 3 + j) << 2));
    return hash_value;
}
int
state_get_hvalue(State state)
{
    return state->h_value;
}

int
state_get_depth(State state)
{
    return state->depth;
}

static void
state_dump(State state)
{
    elog("LOG(state): depth=%d, h=%d, f=%d, ", state->depth, state->h_value,
         state->depth + state->h_value);
    for (int i = 0; i < STATE_N; ++i)
        elog("%d%c", i == state->i + STATE_WIDTH * state->j
                         ? 0
                         : state->pos[i % STATE_WIDTH][i / STATE_WIDTH],
             i == STATE_N - 1 ? '\n' : ',');
}

#include <stddef.h>
#include <stdint.h>
#include <string.h>
#ifndef SIZE_MAX
#define SIZE_MAX ((size_t) -1)
#endif

typedef enum {
    HT_SUCCESS = 0,
    HT_FAILED_FOUND,
    HT_FAILED_NOT_FOUND,
} HTStatus;

/* XXX: hash function for State should be surveyed */
inline static size_t
hashfunc(State key)
{
    return state_hash(key);
}

typedef struct ht_entry_tag *HTEntry;
struct ht_entry_tag
{
    HTEntry next;
    State   key;
    int     value;
};

static HTEntry
ht_entry_init(State key)
{
    HTEntry entry = (HTEntry) palloc(sizeof(*entry));

    entry->key  = state_copy(key);
    entry->next = NULL;

    return entry;
}

static void
ht_entry_fini(HTEntry entry)
{
    pfree(entry);
}

typedef struct ht_tag
{
    size_t   n_bins;
    size_t   n_elems;
    HTEntry *bin;
} * HT;

static bool
ht_rehash_required(HT ht)
{
    return ht->n_bins <= ht->n_elems;
}

static size_t
calc_n_bins(size_t required)
{
    /* NOTE: n_bins is used for mask and hence it should be pow of 2, fon now */
    size_t size = 1;
    assert(required > 0);

    while (required > size)
        size <<= 1;

    return size;
}

HT
ht_init(size_t init_size_hint)
{
    size_t n_bins = calc_n_bins(init_size_hint);
    HT     ht     = (HT) palloc(sizeof(*ht));

    ht->n_bins  = n_bins;
    ht->n_elems = 0;

    assert(sizeof(*ht->bin) <= SIZE_MAX / n_bins);
    ht->bin = (HTEntry *) palloc(sizeof(*ht->bin) * n_bins);
    memset(ht->bin, 0, sizeof(*ht->bin) * n_bins);

    return ht;
}

static void
ht_rehash(HT ht)
{
    HTEntry *new_bin;
    size_t   new_size = ht->n_bins << 1;

    assert(ht->n_bins<SIZE_MAX>> 1);

    new_bin = (HTEntry *) palloc(sizeof(*new_bin) * new_size);
    memset(new_bin, 0, sizeof(*new_bin) * new_size);

    for (size_t i = 0; i < ht->n_bins; ++i)
    {
        HTEntry entry = ht->bin[i];

        while (entry)
        {
            HTEntry next = entry->next;

            size_t idx   = hashfunc(entry->key) & (new_size - 1);
            entry->next  = new_bin[idx];
            new_bin[idx] = entry;

            entry = next;
        }
    }

    pfree(ht->bin);
    ht->n_bins = new_size;
    ht->bin    = new_bin;
}

void
ht_fini(HT ht)
{
    for (size_t i = 0; i < ht->n_bins; ++i)
    {
        HTEntry entry = ht->bin[i];
        while (entry)
        {
            HTEntry next = entry->next;
            state_fini(entry->key);
            ht_entry_fini(entry);
            entry = next;
        }
    }

    pfree(ht->bin);
    pfree(ht);
}

HTStatus
ht_insert(HT ht, State key, int **value)
{
    size_t  i;
    HTEntry entry, new_entry;

    if (ht_rehash_required(ht))
        ht_rehash(ht);

    i     = hashfunc(key) & (ht->n_bins - 1);
    entry = ht->bin[i];

    while (entry)
    {
        if (state_pos_equal(key, entry->key))
        {
            *value = &entry->value;
            return HT_FAILED_FOUND;
        }

        entry = entry->next;
    }

    new_entry = ht_entry_init(key);

    new_entry->next = ht->bin[i];
    ht->bin[i]      = new_entry;
    *value          = &new_entry->value;

    assert(ht->n_elems < SIZE_MAX);
    ht->n_elems++;

    return HT_SUCCESS;
}

/*
 * Priority Queue implementation
 */

#include <assert.h>
#include <stdint.h>

typedef struct pq_entry_tag
{
    State state;
    int   f, g;
} PQEntryData;
typedef PQEntryData *PQEntry;

/* tiebreaking is done comparing g value */
static inline bool
pq_entry_higher_priority(PQEntry e1, PQEntry e2)
{
    return e1->f < e2->f || (e1->f == e2->f && e1->g >= e2->g);
}

/*
 * NOTE:
 * This priority queue is implemented doubly reallocated array.
 * It will only extend and will not shrink, for now.
 * It may be improved by using array of layers of iteratively widened array
 */
typedef struct pq_tag
{
    size_t       n_elems;
    size_t       capa;
    PQEntryData *array;
} * PQ;

static inline size_t
calc_init_capa(size_t capa_hint)
{
    size_t capa = 1;
    assert(capa_hint > 0);

    while (capa < capa_hint)
        capa <<= 1;
    return capa - 1;
}

PQ
pq_init(size_t init_capa_hint)
{
    PQ pq = (PQ) palloc(sizeof(*pq));

    pq->n_elems = 0;
    pq->capa    = calc_init_capa(init_capa_hint);

    assert(pq->capa <= SIZE_MAX / sizeof(PQEntryData));
    pq->array = (PQEntryData *) palloc(sizeof(PQEntryData) * pq->capa);

    return pq;
}

void
pq_fini(PQ pq)
{
    for (size_t i = 0; i < pq->n_elems; ++i)
        state_fini(pq->array[i].state);

    pfree(pq->array);
    pfree(pq);
}

static inline bool
pq_is_full(PQ pq)
{
    assert(pq->n_elems <= pq->capa);
    return pq->n_elems == pq->capa;
}

static inline void
pq_extend(PQ pq)
{
    pq->capa = (pq->capa << 1) + 1;
    assert(pq->capa <= SIZE_MAX / sizeof(PQEntryData));

    pq->array =
        (PQEntryData *) repalloc(pq->array, sizeof(PQEntryData) * pq->capa);
}

static inline void
pq_swap_entry(PQ pq, size_t i, size_t j)
{
    PQEntryData tmp = pq->array[i];
    pq->array[i]    = pq->array[j];
    pq->array[j]    = tmp;
}

static inline size_t
pq_up(size_t i)
{
    /* NOTE: By using 1-origin, it may be written more simply, i >> 1 */
    return (i - 1) >> 1;
}

static inline size_t
pq_left(size_t i)
{
    return (i << 1) + 1;
}

static void
heapify_up(PQ pq)
{
    for (size_t i = pq->n_elems; i > 0;)
    {
        size_t ui = pq_up(i);
        assert(i > 0);
        if (!pq_entry_higher_priority(&pq->array[i], &pq->array[ui]))
            break;

        pq_swap_entry(pq, i, ui);
        i = ui;
    }
}

void
pq_put(PQ pq, State state, int f, int g)
{
    if (pq_is_full(pq))
        pq_extend(pq);

    pq->array[pq->n_elems].state = state_copy(state);
    pq->array[pq->n_elems].f     = f; /* this may be abundant */
    pq->array[pq->n_elems].g     = g;
    heapify_up(pq);
    ++pq->n_elems;
}

static void
heapify_down(PQ pq)
{
    size_t sentinel = pq->n_elems;

    for (size_t i = 0;;)
    {
        size_t ri, li = pq_left(i);
        if (li >= sentinel)
            break;

        ri = li + 1;
        if (ri >= sentinel)
        {
            if (pq_entry_higher_priority(&pq->array[li], &pq->array[i]))
                pq_swap_entry(pq, i, li);
            /* Reached the bottom */
            break;
        }

        /* NOTE: If p(ri) == p(li), it may be good to go right
         * since the filling order is left-first */
        if (pq_entry_higher_priority(&pq->array[li], &pq->array[ri]))
        {
            if (!pq_entry_higher_priority(&pq->array[li], &pq->array[i]))
                break;

            pq_swap_entry(pq, i, li);
            i = li;
        }
        else
        {
            if (!pq_entry_higher_priority(&pq->array[ri], &pq->array[i]))
                break;

            pq_swap_entry(pq, i, ri);
            i = ri;
        }
    }
}

State
pq_pop(PQ pq)
{
    State ret_state;

    if (pq->n_elems == 0)
        return NULL;

    ret_state = pq->array[0].state;

    --pq->n_elems;
    pq->array[0] = pq->array[pq->n_elems];
    heapify_down(pq);

    return ret_state;
}

void
pq_dump(PQ pq)
{
    elog("%s: n_elems=%zu, capa=%zu\n", __func__, pq->n_elems, pq->capa);
    for (size_t i = 0, cr_required = 1; i < pq->n_elems; i++)
    {
        if (i == cr_required)
        {
            elog("\n");
            cr_required = (cr_required << 1) + 1;
        }
        elog("%d,", pq->array[i].f);
        elog("%d ", pq->array[i].g);
    }
    elog("\n");
}

#include <stdlib.h>
#include <string.h>

int
rrand(int m)
{
    return (int) ((double) m * (rand() / (RAND_MAX + 1.0)));
}

void
shuffle_input(Input input[], int n_inputs)
{
    Input  tmp;
    size_t n = n_inputs;
    while (n > 1)
    {
        size_t k = rrand(n--);
        memcpy(&tmp, &input[n], sizeof(Input));
        memcpy(&input[n], &input[k], sizeof(Input));
        memcpy(&input[k], &tmp, sizeof(Input));
    }
}

static HT closed;

bool
distribute_astar(State init_state, Input input[], int distr_n, int *cnt_inputs,
                 int *min_fvalue)
{
    int      cnt = 0;
    State    state;
    PQ       q = pq_init(distr_n + 10);
    HTStatus ht_status;
    int *    ht_value;
    bool     solved = false;
    closed          = ht_init(10000);

    ht_status = ht_insert(closed, init_state, &ht_value);
    *ht_value = 0;
    pq_put(q, state_copy(init_state), state_get_hvalue(init_state), 0);
    ++cnt;

    while ((state = pq_pop(q)))
    {
        --cnt;
        if (state_is_goal(state))
        {
            solved = true;
            break;
        }

        ht_status = ht_insert(closed, state, &ht_value);
        if (ht_status == HT_FAILED_FOUND && *ht_value < state_get_depth(state))
        {
            state_fini(state);
            continue;
        }
        else
            *ht_value = state_get_depth(state);

        for (int dir = 0; dir < DIR_N; ++dir)
        {
            if (state->parent_dir != dir_reverse(dir) &&
                state_movable(state, (Direction) dir))
            {
                State next_state = state_copy(state);
                state_move(next_state, (Direction) dir);
                next_state->depth++;

                ht_status = ht_insert(closed, next_state, &ht_value);
                if (ht_status == HT_FAILED_FOUND &&
                    *ht_value <= state_get_depth(next_state))
                    state_fini(next_state);
                else
                {
                    ++cnt;
                    *ht_value = state_get_depth(next_state);
                    pq_put(q, next_state,
                           *ht_value + state_get_hvalue(next_state), *ht_value);
                }
            }
        }

        state_fini(state);

        if (cnt >= distr_n)
            break;
    }

    *cnt_inputs = cnt;
    elog("LOG: init_distr, cnt=%d\n", cnt);
    if (!solved)
    {
        int minf = INT_MAX;
        for (int id = 0; id < cnt; ++id)
        {
            State state = pq_pop(q);
            assert(state);

            for (int i = 0; i < STATE_N; ++i)
                input[id].tiles[i] =
                    state->pos[i % STATE_WIDTH][i / STATE_WIDTH];
            input[id].tiles[state->i + (state->j * STATE_WIDTH)] = 0;

            input[id].init_depth = state_get_depth(state);
            input[id].parent_dir = state->parent_dir;
            if (minf > state_get_depth(state) + state_get_hvalue(state))
                minf = state_get_depth(state) + state_get_hvalue(state);
        }
        assert(pq_pop(q) == NULL);
        // shuffle_input(input, cnt);
        *min_fvalue = minf;
    }

    pq_fini(q);

    return solved;
}

static int
input_devide(Input input[], search_stat stat[], int i, int devide_n, int tail,
             int *buf_len)
{
    int   cnt = 0;
    int * ht_value;
    State state       = state_init(input[i].tiles, input[i].init_depth);
    state->parent_dir = input[i].parent_dir;
    PQ       pq       = pq_init(devide_n);
    HTStatus ht_status;
    pq_put(pq, state, state_get_hvalue(state), 0);
    ++cnt;
    assert(devide_n > 0);

    while ((state = pq_pop(pq)))
    {
        --cnt;
        if (state_is_goal(state))
        {
            /* It may not be optimal goal */
            pq_put(pq, state, state_get_depth(state) + state_get_hvalue(state),
                   state_get_depth(state));
            ++cnt;
            break;
        }

        ht_status = ht_insert(closed, state, &ht_value);
        if (ht_status == HT_FAILED_FOUND && *ht_value < state_get_depth(state))
        {
            state_fini(state);
            continue;
        }
        else
            *ht_value = state_get_depth(state);

        for (int dir = 0; dir < DIR_N; ++dir)
        {
            if (state->parent_dir != dir_reverse(dir) &&
                state_movable(state, (Direction) dir))
            {
                State next_state = state_copy(state);
                state_move(next_state, (Direction) dir);
                next_state->depth++;

                ht_status = ht_insert(closed, next_state, &ht_value);
                if (ht_status == HT_FAILED_FOUND &&
                    *ht_value < state_get_depth(next_state))
                    state_fini(next_state);
                else
                {
                    ++cnt;
                    *ht_value = state_get_depth(next_state);
                    pq_put(pq, next_state,
                           *ht_value + state_get_hvalue(next_state), *ht_value);
                }
            }
        }

        state_fini(state);

        if (cnt >= devide_n)
            break;
    }

    int new_buf_len = *buf_len;
    while (tail + cnt >= new_buf_len)
        new_buf_len <<= 1;
    if (new_buf_len != *buf_len)
    {
        *buf_len = new_buf_len;
        repalloc(input, sizeof(*input) * new_buf_len);
        elog("LOG: host buf resize\n");
    }

    input[i] = input[tail - 1];

    for (int id = 0; id < cnt; ++id)
    {
        int   ofs   = tail - 1 + id;
        State state = pq_pop(pq);
        assert(state);

        for (int j              = 0; j < STATE_N; ++j)
            input[ofs].tiles[j] = state->pos[j % STATE_WIDTH][j / STATE_WIDTH];
        input[ofs].tiles[state->i + (state->j * STATE_WIDTH)] = 0;

        input[ofs].init_depth = state_get_depth(state);
        input[ofs].parent_dir = state->parent_dir;
    }

    pq_fini(pq);

    return cnt - 1;
}

/* main */

#include <errno.h>
#include <stdio.h>
#include <stdlib.h>

#define exit_failure(...)                                                      \
    do                                                                         \
    {                                                                          \
        printf(__VA_ARGS__);                                                   \
        exit(EXIT_FAILURE);                                                    \
    } while (0)

static int
pop_int_from_str(const char *str, char **end_ptr)
{
    long int rv = strtol(str, end_ptr, 0);
    errno       = 0;

    if (errno != 0)
        exit_failure("%s: %s cannot be converted into long\n", __func__, str);
    else if (end_ptr && str == *end_ptr)
        exit_failure("%s: reach end of string", __func__);

    if (rv > INT_MAX || rv < INT_MIN)
        exit_failure("%s: too big number, %ld\n", __func__, rv);

    return (int) rv;
}

#define MAX_LINE_LEN 100
static void
load_state_from_file(const char *fname, uchar *s)
{
    FILE *fp;
    char  str[MAX_LINE_LEN];
    char *str_ptr = str, *end_ptr;

    fp = fopen(fname, "r");
    if (!fp)
        exit_failure("%s: %s cannot be opened\n", __func__, fname);

    if (!fgets(str, MAX_LINE_LEN, fp))
        exit_failure("%s: fgets failed\n", __func__);

    for (int i = 0; i < STATE_N; ++i)
    {
        s[i]    = pop_int_from_str(str_ptr, &end_ptr);
        str_ptr = end_ptr;
    }

    fclose(fp);
}
#undef MAX_LINE_LEN

#define CUDA_CHECK(call)                                                       \
    do                                                                         \
    {                                                                          \
        const hipError_t e = call;                                            \
        if (e != hipSuccess)                                                  \
            exit_failure("Error: %s:%d code:%d, reason: %s\n", __FILE__,       \
                         __LINE__, e, hipGetErrorString(e));                  \
    } while (0)

__host__ static void *
cudaPalloc(size_t size)
{
    void *ptr;
    CUDA_CHECK(hipMalloc(&ptr, size));
    return ptr;
}

__host__ static void
cudaPfree(void *ptr)
{
    CUDA_CHECK(hipFree(ptr));
}

#define h_d_t(op, i, dir)                                                      \
    (h_diff_table[(op) *STATE_N * DIR_N + (i) *DIR_N + (dir)])
__host__ static void
init_mdist(signed char h_diff_table[])
{
    for (int opponent = 0; opponent < STATE_N; ++opponent)
    {
        int goal_x = POS_X(opponent), goal_y = POS_Y(opponent);

        for (int i = 0; i < STATE_N; ++i)
        {
            int from_x = POS_X(i), from_y = POS_Y(i);
            for (uchar dir = 0; dir < DIR_N; ++dir)
            {
                if (dir == DIR_LEFT)
                    h_d_t(opponent, i, dir) = goal_x > from_x ? -1 : 1;
                if (dir == DIR_RIGHT)
                    h_d_t(opponent, i, dir) = goal_x < from_x ? -1 : 1;
                if (dir == DIR_UP)
                    h_d_t(opponent, i, dir) = goal_y > from_y ? -1 : 1;
                if (dir == DIR_DOWN)
                    h_d_t(opponent, i, dir) = goal_y < from_y ? -1 : 1;
            }
        }
    }
}
#undef h_d_t

#define m_t(i, d) (movable_table[(i) *DIR_N + (d)])
__host__ static void
init_movable_table(bool movable_table[])
{
    for (int i = 0; i < STATE_N; ++i)
        for (unsigned int d = 0; d < DIR_N; ++d)
        {
            if (d == DIR_RIGHT)
                m_t(i, d) = (POS_X(i) < STATE_WIDTH - 1);
            else if (d == DIR_LEFT)
                m_t(i, d) = (POS_X(i) > 0);
            else if (d == DIR_DOWN)
                m_t(i, d) = (POS_Y(i) < STATE_WIDTH - 1);
            else if (d == DIR_UP)
                m_t(i, d) = (POS_Y(i) > 0);
        }
}
#undef m_t

// static char dir_char[] = {'U', 'R', 'L', 'D'};

#define INPUT_SIZE (sizeof(Input) * buf_len)
#define STAT_SIZE (sizeof(search_stat) * buf_len)
#define MOVABLE_TABLE_SIZE (sizeof(bool) * STATE_N * DIR_N)
#define H_DIFF_TABLE_SIZE (STATE_N * STATE_N * DIR_N)
int
main(int argc, char *argv[])
{
    int n_roots;

    int buf_len = N_INIT_DISTRIBUTION * MAX_BUF_RATIO;

    Input *input                = (Input *) palloc(INPUT_SIZE),
          *d_input              = (Input *) cudaPalloc(INPUT_SIZE);
    search_stat *stat           = (search_stat *) palloc(STAT_SIZE),
                *d_stat         = (search_stat *) cudaPalloc(STAT_SIZE);
    bool *movable_table         = (bool *) palloc(MOVABLE_TABLE_SIZE),
         *d_movable_table       = (bool *) cudaPalloc(MOVABLE_TABLE_SIZE);
    signed char *h_diff_table   = (signed char *) palloc(H_DIFF_TABLE_SIZE),
                *d_h_diff_table = (signed char *) cudaPalloc(H_DIFF_TABLE_SIZE);

    int min_fvalue = 0;

    if (argc != 2)
        exit_failure("usage: bin/cumain <ifname>\n");

    load_state_from_file(argv[1], input[0].tiles);

    {
        State init_state = state_init(input[0].tiles, 0);
        state_dump(init_state);
        if (distribute_astar(init_state, input, N_INIT_DISTRIBUTION, &n_roots,
                             &min_fvalue))
        {
            elog("solution is found by distributor\n");
            goto solution_found;
        }
        state_fini(init_state);
    }

    init_mdist(h_diff_table);
    init_movable_table(movable_table);

    CUDA_CHECK(hipMemcpy(d_movable_table, movable_table, MOVABLE_TABLE_SIZE,
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_h_diff_table, h_diff_table, H_DIFF_TABLE_SIZE,
                          hipMemcpyHostToDevice));

    CUDA_CHECK(hipMemset(d_input, 0, INPUT_SIZE));

    for (uchar f_limit = min_fvalue;; f_limit += 2)
    {
        CUDA_CHECK(hipMemset(d_stat, 0, STAT_SIZE));
        CUDA_CHECK(
            hipMemcpy(d_input, input, INPUT_SIZE, hipMemcpyHostToDevice));

        elog("f_limit=%d\n", (int) f_limit);
        idas_kernel<<<n_roots, BLOCK_DIM>>>(d_input, d_stat, f_limit,
                                            d_h_diff_table, d_movable_table);
        CUDA_CHECK(
            hipGetLastError()); /* asm trap is called when find solution */

        CUDA_CHECK(hipMemcpy(stat, d_stat, STAT_SIZE, hipMemcpyDeviceToHost));

        unsigned long long int loads_sum = 0;
        for (int i = 0; i < n_roots; ++i)
            loads_sum += stat[i].loads;

#ifdef COLLECT_LOG
        elog("STAT: loop\n");
        for (int i = 0; i < n_roots; ++i)
            elog("%lld, ", stat[i].loads);
        putchar('\n');
        elog("STAT: nodes_expanded\n");
        for (int i = 0; i < n_roots; ++i)
            elog("%lld, ", stat[i].nodes_expanded);
        putchar('\n');
        elog("STAT: efficiency\n");
        for (int i = 0; i < n_roots; ++i)
		if (stat[i].loads != 0)
            elog("%lld, ", stat[i].nodes_expanded / stat[i].loads);
        putchar('\n');
#endif

        int                    increased = 0;
        unsigned long long int loads_av  = loads_sum / n_roots;

        int stat_cnt[10] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
        for (int i = 0; i < n_roots; ++i)
        {
            if (stat[i].loads < loads_av)
                stat_cnt[0]++;
            else if (stat[i].loads < 2 * loads_av)
                stat_cnt[1]++;
            else if (stat[i].loads < 4 * loads_av)
                stat_cnt[2]++;
            else if (stat[i].loads < 8 * loads_av)
                stat_cnt[3]++;
            else if (stat[i].loads < 16 * loads_av)
                stat_cnt[4]++;
            else if (stat[i].loads < 32 * loads_av)
                stat_cnt[5]++;
            else if (stat[i].loads < 64 * loads_av)
                stat_cnt[6]++;
            else if (stat[i].loads < 128 * loads_av)
                stat_cnt[7]++;
            else
                stat_cnt[8]++;

            int policy = loads_av == 0 ? stat[i].loads
                                       : (stat[i].loads - 1) / loads_av + 1;

            int buf_len_old = buf_len;
            if (policy > 1 && stat[i].loads > 10)
                increased += input_devide(input, stat, i, policy,
                                          n_roots + increased, &buf_len);

            if (buf_len != buf_len_old)
            {
                elog("XXX: fix MAX_BUF_RATIO\n");
                stat = (search_stat *) repalloc(stat, STAT_SIZE);

                cudaPfree(d_input);
                cudaPfree(d_stat);
                d_input = (Input *) cudaPalloc(INPUT_SIZE);
                d_stat  = (search_stat *) cudaPalloc(STAT_SIZE);
            }
        }

        elog("STAT: loads: sum=%lld, av=%lld\n", loads_sum, loads_av);
        elog("STAT: distr: av=%d, 2av=%d, 4av=%d, 8av=%d, 16av=%d, 32av=%d, "
             "64av=%d, 128av=%d, more=%d\n",
             stat_cnt[0], stat_cnt[1], stat_cnt[2], stat_cnt[3], stat_cnt[4],
             stat_cnt[5], stat_cnt[6], stat_cnt[7], stat_cnt[8]);

        n_roots += increased;
        elog("STAT: n_roots=%d(+%d)\n", n_roots, increased);

#ifdef SEARCH_ALL_THE_BEST
        for (int i = 0; i < n_roots; ++i)
            if (stat[i].solved)
            {
                elog("find all the optimal solution(s), at depth=%d\n", stat[i].len);
                goto solution_found;
            }
#endif

        // shuffle_input(input, n_roots); /* it may not be needed in case of idas_global */
    }

solution_found:
    cudaPfree(d_input);
    cudaPfree(d_stat);
    cudaPfree(d_movable_table);
    cudaPfree(d_h_diff_table);

    CUDA_CHECK(hipDeviceReset());

    pfree(input);
    pfree(stat);
    pfree(movable_table);
    pfree(h_diff_table);

    return 0;
}
