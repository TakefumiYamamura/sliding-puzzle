
#include <hip/hip_runtime.h>
#include <stdbool.h>

#define BLOCK_DIM (32)
#define N_BLOCKS (48)
#define N_WORKERS (N_BLOCKS * BLOCK_DIM)
#define N_INIT_DISTRIBUTION (N_WORKERS * 4)
#define N_INPUTS (N_WORKERS * 8)
#define PLAN_LEN_MAX 64

#define STATE_WIDTH 4
#define STATE_N (STATE_WIDTH * STATE_WIDTH)

typedef unsigned char uchar;
typedef signed char   Direction;
#define dir_reverse(dir) ((Direction)(3 - (dir)))
#define DIR_N 4
#define DIR_FIRST 0
#define DIR_UP 0
#define DIR_RIGHT 1
#define DIR_LEFT 2
#define DIR_DOWN 3

/* stack implementation */

__device__ __shared__ static struct dir_stack_tag
{
    uchar i, j;
	uchar parent_dir;
    int   init_depth;
	int   input_i;
    uchar buf[PLAN_LEN_MAX];
} stack[BLOCK_DIM];

#define STACK (stack[threadIdx.x])

typedef struct search_stat_tag
{
    bool      solved;
    int       len;
    unsigned long long int nodes_expanded;
	int thread;
} search_stat;
typedef struct input_tag
{
    uchar     tiles[STATE_N];
    int       init_depth;
    Direction parent_dir;
} Input;

__device__ static inline void
stack_init(Input input, int input_i)
{
    STACK.i          = 0;
	STACK.j          = 0;
	STACK.input_i    = input_i;
	STACK.parent_dir = input.parent_dir;
    STACK.init_depth = input.init_depth;
}

__device__ static inline void
stack_put(Direction dir)
{
    STACK.buf[STACK.i] = dir;
    ++STACK.i;
}
__device__ static inline bool
stack_is_empty(void)
{
    return STACK.i <= STACK.j;
}
__device__ static inline Direction
stack_pop(void)
{
    --STACK.i;
    return STACK.buf[STACK.i];
}
__device__ static inline Direction
stack_peak(void)
{
    return STACK.buf[STACK.i - 1];
}

/* state implementation */

static char assert_state_width_is_four[STATE_WIDTH == 4 ? 1 : -1];
#define POS_X(pos) ((pos) &3)
#define POS_Y(pos) ((pos) >> 2)

/*
 * goal: [0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15]
 */

__device__ __shared__ static struct state_tag
{
    uchar tile[STATE_N];
    uchar empty;
    uchar h_value; /* ub of h_value is 6*16 */
} state[BLOCK_DIM];

#define STATE_TILE(i) (state[threadIdx.x].tile[(i)])
#define STATE_EMPTY (state[threadIdx.x].empty)
#define STATE_HVALUE (state[threadIdx.x].h_value)
#define distance(i, j) ((i) > (j) ? (i) - (j) : (j) - (i))

#define H_DIFF(opponent, empty, empty_dir)                                     \
    h_diff_table_shared[opponent][empty][empty_dir]
__device__ __shared__ static signed char h_diff_table_shared[STATE_N][STATE_N]
                                                            [DIR_N];

__device__ static void
state_init_hvalue(void)
{
    uchar from_x[STATE_N], from_y[STATE_N];

    STATE_HVALUE = 0;

    for (int i = 0; i < STATE_N; ++i)
    {
        from_x[STATE_TILE(i)] = POS_X(i);
        from_y[STATE_TILE(i)] = POS_Y(i);
    }
    for (int i = 1; i < STATE_N; ++i)
    {
        STATE_HVALUE += distance(from_x[i], POS_X(i));
        STATE_HVALUE += distance(from_y[i], POS_Y(i));
    }
}

__device__ static void
state_tile_fill(Input input)
{
    for (int i = 0; i < STATE_N; ++i)
    {
        if (input.tiles[i] == 0)
            STATE_EMPTY = i;
        STATE_TILE(i)   = input.tiles[i];
    }
}

__device__ static inline bool
state_is_goal(void)
{
    return STATE_HVALUE == 0;
}

__device__ static char assert_direction2
    [DIR_UP == 0 && DIR_RIGHT == 1 && DIR_LEFT == 2 && DIR_DOWN == 3 ? 1 : -1];
__device__ __shared__ static bool movable_table_shared[STATE_N][DIR_N];

__device__ static inline bool
state_movable(Direction dir)
{
    return movable_table_shared[STATE_EMPTY][dir];
}

__device__ static char assert_direction
    [DIR_UP == 0 && DIR_RIGHT == 1 && DIR_LEFT == 2 && DIR_DOWN == 3 ? 1 : -1];
__device__ __constant__ const static int pos_diff_table[DIR_N] = {
    -STATE_WIDTH, 1, -1, +STATE_WIDTH};

__device__ static inline bool
state_move_with_limit(Direction dir, unsigned int f_limit)
{
    int new_empty   = STATE_EMPTY + pos_diff_table[dir];
    int opponent    = STATE_TILE(new_empty);
    int new_h_value = STATE_HVALUE + H_DIFF(opponent, new_empty, dir);

    if (STACK.i + STACK.init_depth + 1 + new_h_value > f_limit)
        return false;

    STATE_HVALUE            = new_h_value;
    STATE_TILE(STATE_EMPTY) = opponent;
    STATE_EMPTY             = new_empty;

    return true;
}

__device__ static inline void
state_move(Direction dir)
{
    int new_empty = STATE_EMPTY + pos_diff_table[dir];
    int opponent  = STATE_TILE(new_empty);

    STATE_HVALUE += H_DIFF(opponent, new_empty, dir);
    STATE_TILE(STATE_EMPTY) = opponent;
    STATE_EMPTY             = new_empty;
}

/*
 * solver implementation
 */

__shared__ unsigned int input_i_shared;
#define thread_running (0)
#define thread_sharing (1)
#define thread_stopping (2)
__shared__ int thread_state[32];

__device__ static bool
get_works(Input *input, uchar *dir)
{
	int tid = threadIdx.x;
	int target = (tid + 1) & 31;
	for (;;)
	{
		uchar old = atomicCAS(&thread_state[target], thread_running, thread_sharing);
		if (old == thread_running)
		{
			int j = stack[target].j;
			if (j == stack[target].i)
			{
				target = (target+1)&31;
				thread_state[target] = thread_running;
				if (target == tid)
					break;
				continue;
			}

			int target_i = stack[target].input_i;
			Input in = input[target_i];

			stack_init(in, target_i);
			state_tile_fill(in);
			state_init_hvalue();

			for (int idx = 0; idx < j; ++idx)
				state_move(stack[target].buf[idx]);

			STACK.parent_dir = j == 0 ? stack[target].parent_dir : stack[target].buf[j - 1];
            *dir = stack[target].buf[j];
			STACK.init_depth += j;
			stack[target].j++;

			thread_state[target] = thread_running;
			thread_state[tid] = thread_running;
			return true;
		}
		else if (old == thread_stopping)
		{
			target = (target+1)&31;
			if (target == tid)
				break;
		}
		return false;
	}
	return false;
}

__device__ static void
idas_internal(int f_limit, Input *input, int *input_ends, search_stat *stat)
{
    int       tid            = threadIdx.x;
    int       bid            = blockIdx.x;
    int       id             = tid + bid * blockDim.x;

	int input_begin = bid == 0 ? 0 : input_ends[bid-1];
	int input_end = input_ends[bid];
	int input_i = input_begin+tid;
	uchar     dir            = 0;
	thread_state[tid] = thread_running;

	STACK.input_i = input_i;
	if (input_begin == input_end)
	{
		thread_state[tid] = thread_stopping;
		if (!get_works(input, &dir))
			return;
	}

	/* input surely includes more warks than #warp by devision condition */
	if (tid == 0)
		input_i_shared = input_begin + 32;
	Input this_input = input[input_i];

	stack_init(this_input, input_i);
	state_tile_fill(this_input);
	state_init_hvalue();

	for (;;)
    {
		unsigned long long nodes_expanded = 0;

		for (;;)
		{
			if (state_is_goal())
				asm("trap;"); /* solution found */
			/*
			{
				stat[input_i].solved = true;
				// copy stack to output
				stat[input_i].len = STACK.i;;
				return;
			}
			*/

			if (((stack_is_empty() && dir_reverse(dir) != STACK.parent_dir) ||
						stack_peak() != dir_reverse(dir)) &&
					state_movable(dir))
			{
				++nodes_expanded;

				if (state_move_with_limit(dir, f_limit))
				{
					stack_put(dir);
					dir = 0;
					continue;
				}
			}

			while (++dir == DIR_N)
			{
				if (stack_is_empty())
					goto END_THIS_NODE;

				dir = stack_pop();
				state_move(dir_reverse(dir));
			}
		}

END_THIS_NODE:
        atomicAdd(&stat[input_i].nodes_expanded, nodes_expanded);
        stat[input_i].thread = id; /* just a reference, so not atomic for now */

		input_i = atomicInc(&input_i_shared, UINT_MAX);
		//input_i = ++input_i_shared; /* avoiding atomic operation may improve performance */

		if (input_i >= input_end)
		{
			thread_state[tid] = thread_stopping;
			if (get_works(input, &dir))
			{
				continue;
			}
			else
				return;
		}

		this_input = input[input_i];
		dir            = 0;
		stack_init(this_input, input_i);
		state_tile_fill(this_input);
		state_init_hvalue();
    }
}

__global__ void
idas_kernel(Input *input, int *input_ends, signed char *plan, search_stat *stat,
            int f_limit, signed char *h_diff_table, bool *movable_table)
{
    int       tid            = threadIdx.x;

    for (int dir = 0; dir < DIR_N; ++dir)
        for (int i = tid; i < STATE_N; i += blockDim.x)
            if (i < STATE_N)
                movable_table_shared[i][dir] = movable_table[i * DIR_N + dir];
    for (int i = 0; i < STATE_N * DIR_N; ++i)
        for (int j = tid; j < STATE_N; j += blockDim.x)
            if (j < STATE_N)
                h_diff_table_shared[j][i / DIR_N][i % DIR_N] =
                    h_diff_table[j * STATE_N * DIR_N + i];

    __syncthreads();

	idas_internal(f_limit, input, input_ends, stat);
}

/* host library implementation */

#include <errno.h>
#include <limits.h>
#include <stddef.h>
#include <stdio.h>
#include <stdlib.h>

#ifndef UNABLE_LOG
#define elog(...) fprintf(stderr, __VA_ARGS__)
#else
#define elog(...) ;
#endif

void *
palloc(size_t size)
{
    void *ptr = malloc(size);
    if (!ptr)
        elog("malloc failed\n");

    return ptr;
}

void *
repalloc(void *old_ptr, size_t new_size)
{
    void *ptr = realloc(old_ptr, new_size);
    if (!ptr)
        elog("realloc failed\n");

    return ptr;
}

void
pfree(void *ptr)
{
    if (!ptr)
        elog("empty ptr\n");
    free(ptr);
}

#include <assert.h>
#include <stdbool.h>
#include <stdlib.h>
#include <string.h>

typedef unsigned char idx_t;
/*
 *  [0,0] [1,0] [2,0] [3,0]
 *  [0,1] [1,1] [2,1] [3,1]
 *  [0,2] [1,2] [2,2] [3,2]
 *  [0,3] [1,3] [2,3] [3,3]
 */

/*
 * goal state is
 * [0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15]
 */

typedef struct state_tag_cpu
{
    int       depth; /* XXX: needed? */
    uchar     pos[STATE_WIDTH][STATE_WIDTH];
    idx_t     i, j; /* pos of empty */
    Direction parent_dir;
    int       h_value;
} * State;

#define v(state, i, j) ((state)->pos[i][j])
#define ev(state) (v(state, state->i, state->j))
#define lv(state) (v(state, state->i - 1, state->j))
#define dv(state) (v(state, state->i, state->j + 1))
#define rv(state) (v(state, state->i + 1, state->j))
#define uv(state) (v(state, state->i, state->j - 1))

static uchar from_x[STATE_WIDTH * STATE_WIDTH],
    from_y[STATE_WIDTH * STATE_WIDTH];

static inline void
fill_from_xy(State from)
{
    for (idx_t x = 0; x < STATE_WIDTH; ++x)
        for (idx_t y = 0; y < STATE_WIDTH; ++y)
        {
            from_x[v(from, x, y)] = x;
            from_y[v(from, x, y)] = y;
        }
}

static char assert_state_width_is_four2[STATE_WIDTH == 4 ? 1 : -1];
static inline int
heuristic_manhattan_distance(State from)
{
    int h_value = 0;

    fill_from_xy(from);

    for (idx_t i = 1; i < STATE_N; ++i)
    {
        h_value += distance(from_x[i], i & 3);
        h_value += distance(from_y[i], i >> 2);
    }

    return h_value;
}

bool
state_is_goal(State state)
{
    return state->h_value == 0;
}

static inline State
state_alloc(void)
{
    return (State) palloc(sizeof(struct state_tag_cpu));
}

static inline void
state_free(State state)
{
    pfree(state);
}

State
state_init(uchar v_list[STATE_WIDTH * STATE_WIDTH], int init_depth)
{
    State state = state_alloc();
    int   cnt   = 0;

    state->depth      = init_depth;
    state->parent_dir = (Direction) -1;

    for (idx_t j = 0; j < STATE_WIDTH; ++j)
        for (idx_t i = 0; i < STATE_WIDTH; ++i)
        {
            if (v_list[cnt] == 0)
            {
                state->i = i;
                state->j = j;
            }
            v(state, i, j) = v_list[cnt++];
        }

    state->h_value = heuristic_manhattan_distance(state);

    return state;
}

void
state_fini(State state)
{
    state_free(state);
}

State
state_copy(State src)
{
    State dst = state_alloc();

    memcpy(dst, src, sizeof(*src));

    return dst;
}

static inline bool
state_left_movable(State state)
{
    return state->i != 0;
}
static inline bool
state_down_movable(State state)
{
    return state->j != STATE_WIDTH - 1;
}
static inline bool
state_right_movable(State state)
{
    return state->i != STATE_WIDTH - 1;
}
static inline bool
state_up_movable(State state)
{
    return state->j != 0;
}

bool
state_movable(State state, Direction dir)
{
    return (dir != DIR_LEFT || state_left_movable(state)) &&
           (dir != DIR_DOWN || state_down_movable(state)) &&
           (dir != DIR_RIGHT || state_right_movable(state)) &&
           (dir != DIR_UP || state_up_movable(state));
}

#define h_diff(who, from_i, from_j, dir)                                       \
    (h_diff_table[((who) << 6) + ((from_j) << 4) + ((from_i) << 2) + (dir)])
static int h_diff_table[STATE_N * STATE_N * DIR_N] = {
    1,  1,  1,  1,  1,  1,  -1, 1,  1,  1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  1,
    1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  1,  1,  -1, 1,
    -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  -1,
    1,  -1, 1,  -1, 1,  -1, 1,  1,  -1, 1,  1,  1,  1,  1,  1,  1,  1,  -1, 1,
    1,  1,  -1, 1,  -1, -1, 1,  1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  -1, 1,  -1,
    1,  -1, -1, 1,  1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, -1,
    1,  1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  1,  -1, 1,  1,  1,
    -1, 1,  1,  1,  1,  1,  1,  1,  1,  -1, 1,  -1, -1, 1,  1,  -1, -1, 1,  1,
    -1, 1,  1,  1,  -1, 1,  -1, 1,  -1, -1, 1,  1,  -1, -1, 1,  1,  -1, 1,  1,
    1,  -1, 1,  -1, 1,  -1, -1, 1,  1,  -1, -1, 1,  1,  -1, 1,  1,  1,  -1, 1,
    -1, 1,  1,  -1, 1,  1,  1,  -1, 1,  1,  1,  -1, 1,  1,  1,  1,  1,  1,  -1,
    -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,  -1, 1,  1,  1,  -1, -1, 1,  1,
    -1, -1, 1,  1,  -1, -1, 1,  1,  -1, 1,  1,  1,  -1, -1, 1,  1,  -1, -1, 1,
    1,  -1, -1, 1,  1,  -1, 1,  1,  1,  1,  1,  1,  -1, 1,  1,  -1, -1, 1,  1,
    -1, -1, 1,  1,  -1, -1, 1,  1,  1,  1,  1,  1,  -1, 1,  1,  1,  -1, 1,  1,
    1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,
    -1, 1,  1,  1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  1,  -1, 1,
    -1, 1,  1,  1,  -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,  -1, 1,  1,  1,  1,
    1,  1,  1,  1,  -1, 1,  1,  1,  -1, 1,  -1, -1, 1,  1,  -1, 1,  1,  1,  -1,
    1,  -1, 1,  -1, 1,  -1, 1,  -1, -1, 1,  1,  -1, 1,  1,  1,  -1, 1,  -1, 1,
    -1, 1,  -1, 1,  1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  1,  -1,
    -1, 1,  -1, 1,  1,  1,  -1, 1,  1,  1,  1,  1,  1,  1,  1,  -1, 1,  -1, -1,
    1,  1,  -1, -1, 1,  1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  -1, -1, 1,  1,  -1,
    -1, 1,  1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  1,  -1, 1,  -1, 1,  -1, 1,  -1,
    1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  1,  1,  -1, 1,
    1,  1,  1,  1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,  -1, 1,
    1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,  -1, 1,  1,  1,  1,
    1,  1,  -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,  1,  -1,
    1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,  1,  1,  1,  1,  -1,
    1,  1,  1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  -1, 1,
    -1, 1,  -1, 1,  -1, 1,  1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  1,  -1, -1, 1,
    1,  -1, -1, 1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  1,  -1, -1, 1,  1,  -1, -1,
    1,  -1, 1,  1,  1,  1,  1,  1,  1,  1,  -1, 1,  1,  1,  -1, 1,  -1, -1, 1,
    1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  1,  -1, 1,  -1, 1,  -1,
    1,  -1, 1,  1,  1,  -1, 1,  1,  -1, -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,
    1,  1,  -1, 1,  1,  -1, -1, 1,  -1, 1,  1,  1,  -1, 1,  1,  1,  1,  1,  1,
    1,  1,  -1, 1,  -1, -1, 1,  1,  -1, -1, 1,  1,  -1, 1,  1,  1,  -1, 1,  -1,
    1,  1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  -1,
    1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  1,  1,
    -1, 1,  1,  1,  -1, 1,  1,  1,  1,  1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,
    -1, -1, 1,  1,  -1, 1,  1,  1,  1,  1,  1,  -1, 1,  1,  -1, -1, 1,  1,  -1,
    -1, 1,  1,  -1, -1, 1,  1,  1,  -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,
    -1, -1, 1,  1,  1,  -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,
    1,  1,  1,  1,  1,  -1, 1,  1,  1,  -1, 1,  1,  1,  -1, 1,  1,  -1, 1,  -1,
    1,  1,  1,  -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,  -1, 1,  -1, 1,  1,  1,
    -1, 1,  1,  -1, -1, 1,  1,  -1, -1, 1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  1,
    -1, -1, 1,  1,  -1, -1, 1,  -1, 1,  1,  1,  1,  1,  1,  1,  1,  -1, 1,  1,
    1,  -1, 1,  1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  1,  -1, -1,
    1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  1,  -1, -1, 1,  -1, 1,
    -1, 1,  -1, 1,  -1, 1,  1,  1,  -1, 1,  1,  -1, -1, 1,  -1, 1,  1,  1,  -1,
    1,  1,  1,  1,  1,  1,  1,  1,  -1, 1,  1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,
    -1, 1,  -1, 1,  1,  1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1,
    1,  1,  1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  -1, 1,  1,  1,
    -1, 1,  -1, 1,  1,  1,  -1, 1,  1,  1,  -1, 1,  1,  1,  1,  1,  1};

void
state_move(State state, Direction dir)
{
    idx_t who;
    assert(state_movable(state, dir));

    switch (dir)
    {
    case DIR_LEFT:
        who = ev(state) = lv(state);
        state->i--;
        break;
    case DIR_DOWN:
        who = ev(state) = dv(state);
        state->j++;
        break;
    case DIR_RIGHT:
        who = ev(state) = rv(state);
        state->i++;
        break;
    case DIR_UP:
        who = ev(state) = uv(state);
        state->j--;
        break;
    default:
        elog("unexpected direction");
        assert(false);
    }

    state->h_value =
        state->h_value + h_diff(who, state->i, state->j, dir_reverse(dir));
    state->parent_dir = dir;
}

bool
state_pos_equal(State s1, State s2)
{
    for (idx_t i = 0; i < STATE_WIDTH; ++i)
        for (idx_t j = 0; j < STATE_WIDTH; ++j)
            if (v(s1, i, j) != v(s2, i, j))
                return false;

    return true;
}

size_t
state_hash(State state)
{
    /* FIXME: for A* */
    size_t hash_value = 0;
    for (idx_t i = 0; i < STATE_WIDTH; ++i)
        for (idx_t j = 0; j < STATE_WIDTH; ++j)
            hash_value ^= (v(state, i, j) << ((i * 3 + j) << 2));
    return hash_value;
}
int
state_get_hvalue(State state)
{
    return state->h_value;
}

int
state_get_depth(State state)
{
    return state->depth;
}

#include <stddef.h>
#include <stdint.h>
#include <string.h>
#ifndef SIZE_MAX
#define SIZE_MAX ((size_t) -1)
#endif

typedef enum {
    HT_SUCCESS = 0,
    HT_FAILED_FOUND,
    HT_FAILED_NOT_FOUND,
} HTStatus;

/* XXX: hash function for State should be surveyed */
inline static size_t
hashfunc(State key)
{
    return state_hash(key);
}

typedef struct ht_entry_tag *HTEntry;
struct ht_entry_tag
{
    HTEntry next;
    State   key;
    int     value;
};

static HTEntry
ht_entry_init(State key)
{
    HTEntry entry = (HTEntry) palloc(sizeof(*entry));

    entry->key  = state_copy(key);
    entry->next = NULL;

    return entry;
}

static void
ht_entry_fini(HTEntry entry)
{
    pfree(entry);
}

typedef struct ht_tag
{
    size_t   n_bins;
    size_t   n_elems;
    HTEntry *bin;
} * HT;

static bool
ht_rehash_required(HT ht)
{
    return ht->n_bins <= ht->n_elems; /* TODO: local policy is also needed */
}

static size_t
calc_n_bins(size_t required)
{
    /* NOTE: n_bins is used for mask and hence it should be pow of 2, fon now */
    size_t size = 1;
    assert(required > 0);

    while (required > size)
        size <<= 1;

    return size;
}

HT
ht_init(size_t init_size_hint)
{
    size_t n_bins = calc_n_bins(init_size_hint);
    HT     ht     = (HT) palloc(sizeof(*ht));

    ht->n_bins  = n_bins;
    ht->n_elems = 0;

    assert(sizeof(*ht->bin) <= SIZE_MAX / n_bins);
    ht->bin = (HTEntry *) palloc(sizeof(*ht->bin) * n_bins);
    memset(ht->bin, 0, sizeof(*ht->bin) * n_bins);

    return ht;
}

static void
ht_rehash(HT ht)
{
    HTEntry *new_bin;
    size_t   new_size = ht->n_bins << 1;

    assert(ht->n_bins<SIZE_MAX>> 1);

    new_bin = (HTEntry *) palloc(sizeof(*new_bin) * new_size);
    memset(new_bin, 0, sizeof(*new_bin) * new_size);

    for (size_t i = 0; i < ht->n_bins; ++i)
    {
        HTEntry entry = ht->bin[i];

        while (entry)
        {
            HTEntry next = entry->next;

            size_t idx   = hashfunc(entry->key) & (new_size - 1);
            entry->next  = new_bin[idx];
            new_bin[idx] = entry;

            entry = next;
        }
    }

    pfree(ht->bin);
    ht->n_bins = new_size;
    ht->bin    = new_bin;
}

void
ht_fini(HT ht)
{
    for (size_t i = 0; i < ht->n_bins; ++i)
    {
        HTEntry entry = ht->bin[i];
        while (entry)
        {
            HTEntry next = entry->next;
            state_fini(entry->key);
            ht_entry_fini(entry);
            entry = next;
        }
    }

    pfree(ht->bin);
    pfree(ht);
}

HTStatus
ht_insert(HT ht, State key, int **value)
{
    size_t  i;
    HTEntry entry, new_entry;

    if (ht_rehash_required(ht))
        ht_rehash(ht);

    i     = hashfunc(key) & (ht->n_bins - 1);
    entry = ht->bin[i];

    while (entry)
    {
        if (state_pos_equal(key, entry->key))
        {
            *value = &entry->value;
            return HT_FAILED_FOUND;
        }

        entry = entry->next;
    }

    new_entry = ht_entry_init(key);

    new_entry->next = ht->bin[i];
    ht->bin[i]      = new_entry;
    *value          = &new_entry->value;

    assert(ht->n_elems < SIZE_MAX);
    ht->n_elems++;

    return HT_SUCCESS;
}

/*
 * Priority Queue implementation
 */

#include <assert.h>
#include <stdint.h>

typedef struct pq_entry_tag
{
    State state;
    int   f, g;
} PQEntryData;
typedef PQEntryData *PQEntry;

/* tiebreaking is done comparing g value */
static inline bool
pq_entry_higher_priority(PQEntry e1, PQEntry e2)
{
    return e1->f < e2->f || (e1->f == e2->f && e1->g >= e2->g);
}

/*
 * NOTE:
 * This priority queue is implemented doubly reallocated array.
 * It will only extend and will not shrink, for now.
 * It may be improved by using array of layers of iteratively widened array
 */
typedef struct pq_tag
{
    size_t       n_elems;
    size_t       capa;
    PQEntryData *array;
} * PQ;

static inline size_t
calc_init_capa(size_t capa_hint)
{
    size_t capa = 1;
    assert(capa_hint > 0);

    while (capa < capa_hint)
        capa <<= 1;
    return capa - 1;
}

PQ
pq_init(size_t init_capa_hint)
{
    PQ pq = (PQ) palloc(sizeof(*pq));

    pq->n_elems = 0;
    pq->capa    = calc_init_capa(init_capa_hint);

    assert(pq->capa <= SIZE_MAX / sizeof(PQEntryData));
    pq->array = (PQEntryData *) palloc(sizeof(PQEntryData) * pq->capa);

    return pq;
}

void
pq_fini(PQ pq)
{
    for (size_t i = 0; i < pq->n_elems; ++i)
        state_fini(pq->array[i].state);

    pfree(pq->array);
    pfree(pq);
}

static inline bool
pq_is_full(PQ pq)
{
    assert(pq->n_elems <= pq->capa);
    return pq->n_elems == pq->capa;
}

static inline void
pq_extend(PQ pq)
{
    pq->capa = (pq->capa << 1) + 1;
    assert(pq->capa <= SIZE_MAX / sizeof(PQEntryData));

    pq->array =
        (PQEntryData *) repalloc(pq->array, sizeof(PQEntryData) * pq->capa);
}

static inline void
pq_swap_entry(PQ pq, size_t i, size_t j)
{
    PQEntryData tmp = pq->array[i];
    pq->array[i]    = pq->array[j];
    pq->array[j]    = tmp;
}

static inline size_t
pq_up(size_t i)
{
    /* NOTE: By using 1-origin, it may be written more simply, i >> 1 */
    return (i - 1) >> 1;
}

static inline size_t
pq_left(size_t i)
{
    return (i << 1) + 1;
}

static void
heapify_up(PQ pq)
{
    for (size_t i = pq->n_elems; i > 0;)
    {
        size_t ui = pq_up(i);
        assert(i > 0);
        if (!pq_entry_higher_priority(&pq->array[i], &pq->array[ui]))
            break;

        pq_swap_entry(pq, i, ui);
        i = ui;
    }
}

void
pq_put(PQ pq, State state, int f, int g)
{
    if (pq_is_full(pq))
        pq_extend(pq);

    pq->array[pq->n_elems].state = state_copy(state);
    pq->array[pq->n_elems].f     = f; /* this may be abundant */
    pq->array[pq->n_elems].g     = g;
    heapify_up(pq);
    ++pq->n_elems;
}

static void
heapify_down(PQ pq)
{
    size_t sentinel = pq->n_elems;

    for (size_t i = 0;;)
    {
        size_t ri, li = pq_left(i);
        if (li >= sentinel)
            break;

        ri = li + 1;
        if (ri >= sentinel)
        {
            if (pq_entry_higher_priority(&pq->array[li], &pq->array[i]))
                pq_swap_entry(pq, i, li);
            /* Reached the bottom */
            break;
        }

        /* NOTE: If p(ri) == p(li), it may be good to go right
         * since the filling order is left-first */
        if (pq_entry_higher_priority(&pq->array[li], &pq->array[ri]))
        {
            if (!pq_entry_higher_priority(&pq->array[li], &pq->array[i]))
                break;

            pq_swap_entry(pq, i, li);
            i = li;
        }
        else
        {
            if (!pq_entry_higher_priority(&pq->array[ri], &pq->array[i]))
                break;

            pq_swap_entry(pq, i, ri);
            i = ri;
        }
    }
}

State
pq_pop(PQ pq)
{
    State ret_state;

    if (pq->n_elems == 0)
        return NULL;

    ret_state = pq->array[0].state;

    --pq->n_elems;
    pq->array[0] = pq->array[pq->n_elems];
    heapify_down(pq);

    return ret_state;
}

void
pq_dump(PQ pq)
{
    elog("%s: n_elems=%zu, capa=%zu\n", __func__, pq->n_elems, pq->capa);
    for (size_t i = 0, cr_required = 1; i < pq->n_elems; i++)
    {
        if (i == cr_required)
        {
            elog("\n");
            cr_required = (cr_required << 1) + 1;
        }
        elog("%d,", pq->array[i].f);
        elog("%d ", pq->array[i].g);
    }
    elog("\n");
}

#include <stdlib.h>
#include <string.h>

int rrand(int m)
{
	return (int)((double)m * ( rand() / (RAND_MAX+1.0) ));
}

void shuffle_input(Input input[], search_stat stat[], int n_inputs)
{
	Input tmp;
	search_stat tmp_stat;
	size_t n = n_inputs;
	while ( n > 1 ) {
		size_t k = rrand(n--);

		memcpy(&tmp, &input[n], sizeof(Input));
		memcpy(&input[n], &input[k], sizeof(Input));
		memcpy(&input[k], &tmp, sizeof(Input));

		if (stat)
		{
			memcpy(&tmp_stat, &stat[n], sizeof(Input));
			memcpy(&stat[n], &stat[k], sizeof(Input));
			memcpy(&stat[k], &tmp_stat, sizeof(Input));
		}
	}
}

static HT closed;

bool
distribute_astar(State init_state, Input input[], int input_ends[], int distr_n,
                 int *cnt_inputs, int *min_fvalue)
{
    int      cnt = 0;
    State    state;
    PQ       q = pq_init(distr_n + 10);
    HTStatus ht_status;
    int *    ht_value;
    bool     solved = false;
    closed          = ht_init(10000);

    ht_status = ht_insert(closed, init_state, &ht_value);
    *ht_value = 0;
    pq_put(q, state_copy(init_state), state_get_hvalue(init_state), 0);
    ++cnt;

    while ((state = pq_pop(q)))
    {
        --cnt;
        if (state_is_goal(state))
        {
            solved = true;
            break;
        }

        ht_status = ht_insert(closed, state, &ht_value);
        if (ht_status == HT_FAILED_FOUND && *ht_value < state_get_depth(state))
        {
            state_fini(state);
            continue;
        }
        else
            *ht_value = state_get_depth(state);

        for (int dir = 0; dir < DIR_N; ++dir)
        {
            if (state->parent_dir != dir_reverse(dir) &&
                state_movable(state, (Direction) dir))
            {
                State next_state = state_copy(state);
                state_move(next_state, (Direction) dir);
                next_state->depth++;

                ht_status = ht_insert(closed, next_state, &ht_value);
                if (ht_status == HT_FAILED_FOUND &&
                    *ht_value <= state_get_depth(next_state))
                    state_fini(next_state);
                else
                {
                    ++cnt;
                    *ht_value = state_get_depth(next_state);
                    pq_put(q, next_state,
                           *ht_value + state_get_hvalue(next_state), *ht_value);
                }
            }
        }

        state_fini(state);

        if (cnt >= distr_n)
            break;
    }

    *cnt_inputs = cnt;
    if (!solved)
    {
        int minf = INT_MAX;
        for (int id = 0; id < cnt; ++id)
        {
            State state = pq_pop(q);
            assert(state);

            for (int i = 0; i < STATE_N; ++i)
                input[id].tiles[i] =
                    state->pos[i % STATE_WIDTH][i / STATE_WIDTH];
            input[id].tiles[state->i + (state->j * STATE_WIDTH)] = 0;

            input[id].init_depth = state_get_depth(state);
            input[id].parent_dir = state->parent_dir;
            if (minf > state_get_depth(state) + state_get_hvalue(state))
                minf = state_get_depth(state) + state_get_hvalue(state);
        }
		shuffle_input(input, NULL, cnt);
        *min_fvalue = minf;

        printf("distr_n=%d, n_worers=%d, cnt=%d\n", distr_n, N_WORKERS, cnt);
        for (int id               = 0; id < N_BLOCKS; ++id)
            input_ends[id]        = (distr_n / N_BLOCKS) * (id + 1) - 1;
        input_ends[N_BLOCKS - 1] = cnt;
    }

    pq_fini(q);

    return solved;
}

static int
input_devide(Input input[], search_stat stat[], int i, int devide_n, int tail)
{
    int   cnt = 0;
    int * ht_value;
    State state       = state_init(input[i].tiles, input[i].init_depth);
    state->parent_dir = input[i].parent_dir;
    PQ       pq       = pq_init(32);
    HTStatus ht_status;
    pq_put(pq, state, state_get_hvalue(state), 0);
    ++cnt;

    while ((state = pq_pop(pq)))
    {
        --cnt;
        if (state_is_goal(state))
        {
            /* It may not be optimal goal */
            pq_put(pq, state, state_get_depth(state) + state_get_hvalue(state),
                   state_get_depth(state));
            ++cnt;
            break;
        }

        ht_status = ht_insert(closed, state, &ht_value);
        if (ht_status == HT_FAILED_FOUND && *ht_value < state_get_depth(state))
        {
            state_fini(state);
            continue;
        }
        else
            *ht_value = state_get_depth(state);

        for (int dir = 0; dir < DIR_N; ++dir)
        {
            if (state->parent_dir != dir_reverse(dir) &&
                state_movable(state, (Direction) dir))
            {
                State next_state = state_copy(state);
                state_move(next_state, (Direction) dir);
                next_state->depth++;

                ht_status = ht_insert(closed, next_state, &ht_value);
                if (ht_status == HT_FAILED_FOUND &&
                    *ht_value < state_get_depth(next_state))
                    state_fini(next_state);
                else
                {
                    ++cnt;
                    *ht_value = state_get_depth(next_state);
                    pq_put(pq, next_state,
                           *ht_value + state_get_hvalue(next_state), *ht_value);
                }
            }
        }

        state_fini(state);

        if (cnt >= devide_n)
            break;
    }

    for (int id = 0; id < cnt; ++id)
    {
        int   estimation_after_devision = stat[i].nodes_expanded / cnt; /* XXX: fix to consider f-value */
        int   ofs                       = id == 0 ? i : tail - 1 + id;
        State state                     = pq_pop(pq);
        assert(state);

        for (int j              = 0; j < STATE_N; ++j)
            input[ofs].tiles[j] = state->pos[j % STATE_WIDTH][j / STATE_WIDTH];
        input[ofs].tiles[state->i + (state->j * STATE_WIDTH)] = 0;

        input[ofs].init_depth = state_get_depth(state);
        input[ofs].parent_dir = state->parent_dir;

        stat[ofs].nodes_expanded = estimation_after_devision;
    }

    pq_fini(pq);

    return cnt == 0 ? 0 : cnt - 1;
}

/* main */

#include <errno.h>
#include <stdio.h>
#include <stdlib.h>

#define exit_failure(...)                                                      \
    do                                                                         \
    {                                                                          \
        printf(__VA_ARGS__);                                                   \
        exit(EXIT_FAILURE);                                                    \
    } while (0)

static int
pop_int_from_str(const char *str, char **end_ptr)
{
    long int rv = strtol(str, end_ptr, 0);
    errno       = 0;

    if (errno != 0)
        exit_failure("%s: %s cannot be converted into long\n", __func__, str);
    else if (end_ptr && str == *end_ptr)
        exit_failure("%s: reach end of string", __func__);

    if (rv > INT_MAX || rv < INT_MIN)
        exit_failure("%s: too big number, %ld\n", __func__, rv);

    return (int) rv;
}

#define MAX_LINE_LEN 100
static void
load_state_from_file(const char *fname, uchar *s)
{
    FILE *fp;
    char  str[MAX_LINE_LEN];
    char *str_ptr = str, *end_ptr;

    fp = fopen(fname, "r");
    if (!fp)
        exit_failure("%s: %s cannot be opened\n", __func__, fname);

    if (!fgets(str, MAX_LINE_LEN, fp))
        exit_failure("%s: fgets failed\n", __func__);

    for (int i = 0; i < STATE_N; ++i)
    {
        s[i]    = pop_int_from_str(str_ptr, &end_ptr);
        str_ptr = end_ptr;
    }

    fclose(fp);
}
#undef MAX_LINE_LEN

#define CUDA_CHECK(call)                                                       \
    do                                                                         \
    {                                                                          \
        const hipError_t e = call;                                            \
        if (e != hipSuccess)                                                  \
            exit_failure("Error: %s:%d code:%d, reason: %s\n", __FILE__,       \
                         __LINE__, e, hipGetErrorString(e));                  \
    } while (0)

#define h_d_t(op, i, dir)                                                      \
    (h_diff_table[(op) *STATE_N * DIR_N + (i) *DIR_N + (dir)])
__host__ static void
init_mdist(signed char h_diff_table[])
{
    for (int opponent = 0; opponent < STATE_N; ++opponent)
    {
        int goal_x = POS_X(opponent), goal_y = POS_Y(opponent);

        for (int i = 0; i < STATE_N; ++i)
        {
            int from_x = POS_X(i), from_y = POS_Y(i);
            for (uchar dir = 0; dir < DIR_N; ++dir)
            {
                if (dir == DIR_LEFT)
                    h_d_t(opponent, i, dir) = goal_x > from_x ? -1 : 1;
                if (dir == DIR_RIGHT)
                    h_d_t(opponent, i, dir) = goal_x < from_x ? -1 : 1;
                if (dir == DIR_UP)
                    h_d_t(opponent, i, dir) = goal_y > from_y ? -1 : 1;
                if (dir == DIR_DOWN)
                    h_d_t(opponent, i, dir) = goal_y < from_y ? -1 : 1;
            }
        }
    }
}
#undef h_d_t

#define m_t(i, d) (movable_table[(i) *DIR_N + (d)])
__host__ static void
init_movable_table(bool movable_table[])
{
    for (int i = 0; i < STATE_N; ++i)
        for (unsigned int d = 0; d < DIR_N; ++d)
        {
            if (d == DIR_RIGHT)
                m_t(i, d) = (POS_X(i) < STATE_WIDTH - 1);
            else if (d == DIR_LEFT)
                m_t(i, d) = (POS_X(i) > 0);
            else if (d == DIR_DOWN)
                m_t(i, d) = (POS_Y(i) < STATE_WIDTH - 1);
            else if (d == DIR_UP)
                m_t(i, d) = (POS_Y(i) > 0);
        }
}
#undef m_t

static void
avoid_unused_static_assertions(void)
{
    (void) assert_direction[0];
    (void) assert_direction2[0];
    (void) assert_state_width_is_four[0];
    (void) assert_state_width_is_four2[0];
}

static char dir_char[] = {'U', 'R', 'L', 'D'};

int
main(int argc, char *argv[])
{
    int cnt_inputs;

    int    input_size = sizeof(Input) * N_INPUTS;
    Input  input[N_INPUTS];
    Input *d_input;

    int  input_ends_size = sizeof(int) * N_BLOCKS;
    int  input_ends[N_BLOCKS];
    int *d_input_ends;

    int          plan_size = sizeof(signed char) * PLAN_LEN_MAX * N_INPUTS;
    signed char  plan[PLAN_LEN_MAX * N_INPUTS];
    signed char *d_plan;

    int          stat_size = sizeof(search_stat) * N_INPUTS;
    search_stat  stat[N_INPUTS];
    search_stat *d_stat;

    bool         movable_table[STATE_N * DIR_N];
    bool *       d_movable_table;
    int          movable_table_size = sizeof(bool) * STATE_N * DIR_N;
    signed char  h_diff_table[STATE_N * STATE_N * DIR_N];
    signed char *d_h_diff_table;
    int h_diff_table_size = sizeof(signed char) * STATE_N * STATE_N * DIR_N;

    int min_fvalue = 0;

    if (argc < 2)
    {
        printf("usage: bin/cumain <ifname>\n");
        exit(EXIT_FAILURE);
    }

    load_state_from_file(argv[1], input[0].tiles);

    {
        State init_state = state_init(input[0].tiles, 0);

        if (distribute_astar(init_state, input, input_ends, N_INIT_DISTRIBUTION,
                             &cnt_inputs, &min_fvalue))
        {
            puts("solution is found by distributor");
            return 0;
        }
    }

    init_mdist(h_diff_table);
    init_movable_table(movable_table);

    CUDA_CHECK(hipMalloc((void **) &d_input, input_size));
    CUDA_CHECK(hipMalloc((void **) &d_input_ends, input_ends_size));
    CUDA_CHECK(hipMalloc((void **) &d_plan, plan_size));
    CUDA_CHECK(hipMalloc((void **) &d_stat, stat_size));
    CUDA_CHECK(hipMalloc((void **) &d_movable_table, movable_table_size));
    CUDA_CHECK(hipMalloc((void **) &d_h_diff_table, h_diff_table_size));
    CUDA_CHECK(hipMemcpy(d_movable_table, movable_table, movable_table_size,
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_h_diff_table, h_diff_table, h_diff_table_size,
                          hipMemcpyHostToDevice));

    CUDA_CHECK(hipMemset(d_input, 0, input_size));
    CUDA_CHECK(hipMemset(d_plan, 0, plan_size));

    for (uchar f_limit = min_fvalue;; f_limit += 2)
    {
		CUDA_CHECK(hipMemset(d_stat, 0, stat_size));
        elog("f=%d\n", (int) f_limit);

        CUDA_CHECK(
            hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_input_ends, input_ends, input_ends_size,
                              hipMemcpyHostToDevice));

        elog("call idas_kernel(block=%d, thread=%d)\n", N_BLOCKS, BLOCK_DIM);
        idas_kernel<<<N_BLOCKS, BLOCK_DIM>>>(d_input, d_input_ends, d_plan,
                                             d_stat, f_limit, d_h_diff_table,
                                             d_movable_table);
        CUDA_CHECK(hipGetLastError());

        CUDA_CHECK(hipMemcpy(plan, d_plan, plan_size, hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(stat, d_stat, stat_size, hipMemcpyDeviceToHost));

        for (int i = 0; i < cnt_inputs; ++i)
            if (stat[i].solved)
            {
                elog("core id = %d\n", i);
                printf("cpu len=%d: \n", input[i].init_depth);

                /* CPU side output */
                // FIXME: Not implemented, for now. It is easy to search path
                // from init state to this root.

                /* GPU side output */
                printf("gpu len=%d: ", stat[i].len);
                for (int j = 0; j < stat[i].len; ++j)
                    printf("%c ", dir_char[(int) plan[i * PLAN_LEN_MAX + j]]);
                putchar('\n');

                goto solution_found;
            }

		unsigned long long int nodes_expanded_by_threads[N_WORKERS];
		memset(nodes_expanded_by_threads, 0, sizeof(nodes_expanded_by_threads[0]) * N_WORKERS);
        unsigned long long int sum_of_expansion = 0;
        for (int i = 0; i < cnt_inputs; ++i)
		{
            sum_of_expansion += stat[i].nodes_expanded;
			nodes_expanded_by_threads[stat[i].thread] += stat[i].nodes_expanded;
		}

        printf("STAT: nodes_expanded\n");
        for (int i = 0; i < cnt_inputs; ++i)
            printf("%lld, ", stat[i].nodes_expanded);
        putchar('\n');
        printf("STAT: threads_loads\n");
        for (int i = 0; i < N_WORKERS; ++i)
            printf("%lld, ", nodes_expanded_by_threads[i]);
        putchar('\n');

        int increased             = 0;
		unsigned long long int avarage_expected_load = sum_of_expansion / N_WORKERS;

        int stat_cnt[10] = {0, 0, 0, 0, 0, 0, 0};
        for (int i = 0; i < cnt_inputs; ++i)
        {
            if (stat[i].nodes_expanded < avarage_expected_load)
                stat_cnt[0]++;
            else if (stat[i].nodes_expanded < 2 * avarage_expected_load)
                stat_cnt[1]++;
            else if (stat[i].nodes_expanded < 4 * avarage_expected_load)
                stat_cnt[2]++;
            else if (stat[i].nodes_expanded < 8 * avarage_expected_load)
                stat_cnt[3]++;
            else if (stat[i].nodes_expanded < 16 * avarage_expected_load)
                stat_cnt[4]++;
            else if (stat[i].nodes_expanded < 32 * avarage_expected_load)
                stat_cnt[5]++;
            else
                stat_cnt[6]++;

            int policy =
                (stat[i].nodes_expanded - 1)/ avarage_expected_load + 1;

            if (policy > 1 && stat[i].nodes_expanded > 100)
	    {
                increased += input_devide(input, stat, i, policy,
                                          cnt_inputs + increased);
	    }
        }
        elog("STAT: sum of expanded nodes: %lld\n", sum_of_expansion);
        elog("STAT: avarage expanded nodes: %lld\n", avarage_expected_load);
        elog("STAT: av=%d, 2av=%d, 4av=%d, 8av=%d, 16av=%d, 32av=%d, more=%d\n",
             stat_cnt[0], stat_cnt[1], stat_cnt[2], stat_cnt[3], stat_cnt[4],
             stat_cnt[5], stat_cnt[6]);

	elog("DEBUG: cnt_inputs=%d, increased=%d\n", cnt_inputs, increased);

        if (cnt_inputs + increased > N_INPUTS)
        {
            elog("cnt_inputs too large, cnt_inputs=%d\n", cnt_inputs + increased);
            abort();
        }

        cnt_inputs += increased;
        elog("input count: %d\n", cnt_inputs);

        int stat_thread[10] = {0, 0, 0, 0, 0, 0, 0};
        for (int i = 0; i < N_WORKERS; ++i)
        {
            if (nodes_expanded_by_threads[i]< avarage_expected_load)
                stat_thread[0]++;
            else if (nodes_expanded_by_threads[i]< 2 * avarage_expected_load)
                stat_thread[1]++;
            else if (nodes_expanded_by_threads[i]< 4 * avarage_expected_load)
                stat_thread[2]++;
            else if (nodes_expanded_by_threads[i]< 8 * avarage_expected_load)
                stat_thread[3]++;
            else if (nodes_expanded_by_threads[i]< 16 * avarage_expected_load)
                stat_thread[4]++;
            else if (nodes_expanded_by_threads[i]< 32 * avarage_expected_load)
                stat_thread[5]++;
            else
                stat_thread[6]++;
        }
        elog("STAT: avarage thread_wors: %lld\n", avarage_expected_load);
        elog("STAT: av=%d, 2av=%d, 4av=%d, 8av=%d, 16av=%d, 32av=%d, more=%d\n",
             stat_thread[0], stat_thread[1], stat_thread[2], stat_thread[3],
			 stat_thread[4], stat_thread[5], stat_thread[6]);

		shuffle_input(input, stat, cnt_inputs);

        /* NOTE: optionally sort here by expected cost or g/h-value */

        int id = 0;
        for (int i = 0, load = 0; i < cnt_inputs; ++i)
        {
            load += stat[i].nodes_expanded;
            if ((unsigned int) load >= avarage_expected_load*BLOCK_DIM)
            {
                load             = 0;
                input_ends[id++] = i;
            }
        }

        while (id < N_BLOCKS)
            input_ends[id++] = cnt_inputs;
	input_ends[N_BLOCKS-1] = cnt_inputs;
    }
solution_found:

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_input_ends));
    CUDA_CHECK(hipFree(d_plan));
    CUDA_CHECK(hipFree(d_stat));
    CUDA_CHECK(hipFree(d_movable_table));
    CUDA_CHECK(hipFree(d_h_diff_table));
    CUDA_CHECK(hipDeviceReset());

    avoid_unused_static_assertions();

    return 0;
}
